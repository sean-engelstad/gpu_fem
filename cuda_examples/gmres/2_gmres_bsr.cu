#include "hip/hip_runtime.h"
#include "_gmres_utils.h"
#include "_mat_utils.h"

int main() {
    using T = double;

    // case inputs
    // -----------

    constexpr bool test_mult = true;
    constexpr bool check_mat_data = true;
    int N = 4; // 16384
    int n_iter = min(N, 200);
    constexpr bool use_precond = true;
    constexpr bool test_precond = false;
    bool debug = false;
    T abs_tol = 1e-8, rel_tol = 1e-8;

    // NOTE : starting with BSR matrix of block size 1 (just to demonstrate the correct cusparse methods for BSR)

    // initialize data
    // ---------------
    
    int *rowp, *cols;
    // int M = N;
    T *vals, *rhs, *x;
    int nz = 5 * N - 4 * (int)sqrt((double)N);

    // allocate rowp, cols on host
    rowp = (int*)malloc(sizeof(int) * (N + 1));
    cols = (int*)malloc(sizeof(int) * nz);
    vals = (T*)malloc(sizeof(T) * nz);
    x = (T*)malloc(sizeof(T) * N);
    rhs = (T*)malloc(sizeof(T) * N);

    for (int i = 0; i < N; i++) {
        x[i] = 0.0;
        rhs[i] = 0.0;    
    }

    // initialize data
    genLaplaceCSR<T>(rowp, cols, vals, N, nz, rhs);
    // now rhs is not zero

    // transfer data to the device
    int *d_rowp, *d_cols;
    T *d_vals, *d_x, *d_rhs, *d_vals_ILU0;
    CHECK_CUDA(hipMalloc((void **)&d_rowp, (N+1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_cols, nz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_vals, nz * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_vals_ILU0, nz * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_x, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_rhs, N * sizeof(T)));

    // copy data for the matrix over to device
    CHECK_CUDA(hipMemcpy(d_rowp, rowp, (N+1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_cols, cols, nz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_vals, vals, nz * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, N * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_rhs, rhs, N * sizeof(T), hipMemcpyHostToDevice));

    if constexpr (check_mat_data) {
        // also print out d_rowp, d_cols, d_vals to double check their values on host
        int *h_rowp = new int[5];
        int *h_cols = new int[12];
        T *h_vals = new T[12];
        
        CHECK_CUDA(hipMemcpy(h_rowp, d_rowp, 5 * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_cols, d_cols, 12 * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_vals, d_vals, 12 * sizeof(T), hipMemcpyDeviceToHost));
        printf("d_rowp:");
        printVec<int>(5, h_rowp);
        printf("d_cols:");
        printVec<int>(12, h_cols);
        printf("d_vals:");
        printVec<T>(12, h_vals);
    }

    // create temp vec objects
    // -----------------------

    double *d_resid, *d_tmp, *d_w;
    CHECK_CUDA(hipMalloc((void **)&d_resid, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_tmp, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_w, N * sizeof(T)));

    // create initial cusparse and cublas objects
    // ------------------------------------------

    /* Create CUBLAS context */
    hipblasHandle_t cublasHandle = NULL;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    /* Create CUSPARSE context */
    hipsparseHandle_t cusparseHandle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    /* Description of the A matrix */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    // wrap dense vectors into cusparse dense vector objects
    // -----------------------------------------------------

    hipsparseDnVecDescr_t vec_rhs, vec_tmp, vec_w;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_rhs, N, d_rhs, HIP_R_64F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_tmp, N, d_tmp, HIP_R_64F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_w, N, d_w, HIP_R_64F));
    
    // create the matrix BSR objects
    // -----------------------------

    // bsr data
    int mb = N;
    int block_dim = 1;
    int nnzb = nz;


    // create ILU(0) preconditioner
    // ----------------------------


    // first test the matrix-vec product on GPU
    // ----------------------------------------
    
    if constexpr (test_mult) {
        assert(4 == N);
        T *my_vec = new T[4];
        for (int i = 0; i < 4; i++) {
            my_vec[i] = i+1;
        }

        printf("mb = %d, nnzb = %d, block_dim = %d\n", mb, nnzb, block_dim);

        T *d_my_vec;
        CHECK_CUDA(hipMalloc((void **)&d_my_vec, 4 * sizeof(T)));
        CHECK_CUDA(hipMemcpy(d_my_vec, my_vec, 4 * sizeof(T), hipMemcpyHostToDevice));
        hipsparseDnVecDescr_t vec_my_vec = NULL;
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_my_vec, N, d_my_vec, HIP_R_64F));

        printf("here1\n");

        assert(block_dim > 0);
        assert(mb > 0);
        assert(nnzb > 0);
        assert(d_vals != nullptr);
        assert(d_rowp != nullptr);
        assert(d_cols != nullptr);
        assert(d_my_vec != nullptr);
        assert(d_tmp != nullptr);       


        printf("mb = %d, nnzb = %d, block_dim = %d\n", mb, nnzb, block_dim);
        printf("d_vals = %p, d_rowp = %p, d_cols = %p, d_my_vec = %p, d_tmp = %p\n",
            d_vals, d_rowp, d_cols, d_my_vec, d_tmp);


        // A * my_vec => tmp1
        // TODO: BSR MV here
        T a = 1.0, b = 0.0;
        CHECK_CUSPARSE(hipsparseDbsrmv(
            cusparseHandle, 
            HIPSPARSE_DIRECTION_ROW,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            mb, mb, nnzb,
            &a, descrA,
            d_vals, d_rowp, d_cols,
            block_dim,
            d_my_vec,
            &b,
            d_tmp
        ));
        
        printf("here2\n");

        // should get this result in tmp1
        // t1=array([  1,  -3,  -7, -11])

        // copy data back to host x vec (just a test of matmult here)
        CHECK_CUDA(hipMemcpy(x, d_tmp, N * sizeof(T), hipMemcpyDeviceToHost));
        if (debug && N == 4) {
            printf("A*my_vec=");
            printVec<T>(4, x);
        }

        return 0;
    }


    // GMRES solve now with CSR matrix
    // -------------------------------

    printf("checkpt\n");

    // initialize GMRES data, some on host, some on GPU
    // host GMRES data
    T g[n_iter+1], cs[n_iter], ss[n_iter];
    T H[(n_iter+1)*(n_iter)];

    // GMRES device data
    T *d_Vmat, *d_V;
    CHECK_CUDA(hipMalloc((void **)&d_Vmat, (n_iter+1) * N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_V, N * sizeof(T)));
    // use single hipsparseDnVecDescr_t of size N and just update it's values occasionally
    hipsparseDnVecDescr_t vec_V;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_V, N, d_V, HIP_R_64F));
    // other strategy is to make pointer array of hipsparseDnVecDescr_t vecs
    // update with void *col_ptr = static_cast<void*>(&d_Vmat[k * N]);
    //             hipsparseDnVecSetValues(vec_V, col_ptr)

    // setup the ILU(0) preconditioner (if in use)
    // -------------------------------------------

    if constexpr (use_precond) {
        // setup ILU(0) preconditioner here..
    }

    if constexpr (test_precond && use_precond) {
        // print A matrix values::
        T *h_A = new T[nz];
        T *h_M = new T[nz];
        CHECK_CUDA(hipMemcpy(h_A, d_vals, nz * sizeof(T), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_M, d_vals_ILU0, nz * sizeof(T), hipMemcpyDeviceToHost));
        
        printf("h_A:");
        printVec<T>(nz, h_A);
        printf("h_M:");
        printVec<T>(nz, h_M);

        // test LU solve on each unit vector..
    }


    // GMRES algorithm
    // ----------------------------

    int jj = n_iter - 1;

    // apply precond to rhs if in use
    if constexpr (use_precond) {
        // print part of initial vec_rhs
        int NPRINT = N;
        T *h_rhs = new T[NPRINT];
        CHECK_CUDA(hipMemcpy(h_rhs, d_rhs, NPRINT * sizeof(T), hipMemcpyDeviceToHost));
        // printf("init vec_rhs:");
        // printVec<T>(NPRINT, h_rhs);

        // zero vec_tmp
        CHECK_CUDA(hipMemset(d_tmp, 0.0, N * sizeof(T)));
        
        // TODO : ILU solve U^-1 L^-1 * b

        // CHECK_CUDA(hipMemcpy(h_rhs, d_rhs, NPRINT * sizeof(T), hipMemcpyDeviceToHost));
        // printf("precond vec_rhs:");
        // printVec<T>(NPRINT, h_rhs);
    }

    // GMRES initial residual
    // assumes here d_X is 0 initially => so r0 = b - Ax = b
    T beta;
    CHECK_CUBLAS(hipblasDnrm2(cublasHandle, N, d_rhs, 1, &beta));
    printf("GMRES init resid = %.5e\n", beta);
    g[0] = beta;

    // set v0 = r0 / beta (unit vec)
    T a = 1.0 / beta;
    CHECK_CUBLAS(hipblasDaxpy(cublasHandle, N, &a, d_rhs, 1, &d_Vmat[0], 1));

    T *h_v0 = new T[N];
    CHECK_CUDA(hipMemcpy(h_v0, d_Vmat, N * sizeof(T), hipMemcpyDeviceToHost));
    // print vec
    if (debug) {
        printf("r0:");
        printVec<T>(4, h_v0);
    }

    // then begin main GMRES iteration loop!
    for (int j = 0; j < n_iter; j++) {
        // zero this vec
        // CHECK_CUDA(hipMemset(&d_Vmat[j * N], 0.0, N * sizeof(T)));

        // get vj and copy it into the cusparseDnVec_t
        void *vj_col = static_cast<void*>(&d_Vmat[j * N]);
        CHECK_CUSPARSE(hipsparseDnVecSetValues(vec_V, vj_col));

        // w = A * vj + 0 * w
        // TODO : BSR matrix multiply here MV
        // CHECK_CUSPARSE(hipsparseDbsrmv(
        //     cusparseHandle, 
        //     HIPSPARSE_DIRECTION_ROW,
        //     HIPSPARSE_OPERATION_NON_TRANSPOSE,
        //     mb, mb, nnzb,
        //     &a, descrA,
        //     kmat_vals, dk_rowp, dk_cols,
        //     block_dim,
        //     soln_ptr,
        //     &b,
        //     resid_ptr
        // ));

        if constexpr (use_precond) {
            // TODO : U^-1 L^-1 * w => w precond solve here
        }

        // double check and print the value of 
        if (debug && N <= 16) {
            T *h_w = new T[N];
            // printf("checkpt3\n");
            CHECK_CUDA(hipMemcpy(h_w, d_w, N * sizeof(T), hipMemcpyDeviceToHost));
            printf("h_w[%d] pre-GS:", j);
            printVec<T>(N, h_w);

            // if (j == 0) return 0;
        }

        // now update householder matrix
        for (int i = 0 ; i < j+1; i++) {
            // get vi column
            void *vi_col = static_cast<void*>(&d_Vmat[i * N]);
            CHECK_CUSPARSE(hipsparseDnVecSetValues(vec_V, vi_col));

            T w_vi_dot;
            CHECK_CUBLAS(hipblasDdot(cublasHandle, N, d_w, 1, &d_Vmat[i * N], 1, &w_vi_dot));

            // H_ij = vi dot w
            H[n_iter * i + j] = w_vi_dot;

            if (debug) printf("H[%d,%d] = %.4e\n", i, j, H[n_iter * i + j]);
            
            // w -= Hij * vi
            a = -H[n_iter * i + j];
            CHECK_CUBLAS(hipblasDaxpy(cublasHandle, N, &a, &d_Vmat[i * N], 1, d_w, 1));
        }

        // double check and print the value of 
        if (debug && N <= 16) {
            T *h_w = new T[N];
            // printf("checkpt3\n");
            CHECK_CUDA(hipMemcpy(h_w, d_w, N * sizeof(T), hipMemcpyDeviceToHost));
            printf("h_w[%d] post GS:", j);
            printVec<T>(N, h_w);

            // if (j == 0) return 0;
        }

        // norm of w
        T nrm_w;
        CHECK_CUBLAS(hipblasDnrm2(cublasHandle, N, d_w, 1, &nrm_w));

        // H_{j+1,j}
        H[n_iter * (j+1) + j] = nrm_w;

        // v_{j+1} column unit vec = w / H_{j+1,j}
        a = 1.0 / H[n_iter * (j+1) + j];
        CHECK_CUBLAS(hipblasDcopy(cublasHandle, N, d_w, 1, &d_Vmat[(j+1) * N], 1));
        CHECK_CUBLAS(hipblasDscal(cublasHandle, N, &a, &d_Vmat[(j+1) * N], 1));

        if (debug && N <= 16) {
            T *h_tmp = new T[N];
            // printf("checkpt3\n");
            CHECK_CUDA(hipMemcpy(h_tmp, &d_Vmat[(j+1) * N], N * sizeof(T), hipMemcpyDeviceToHost));
            printf("next V:");
            printVec<T>(N, h_tmp);

            // if (j == 0) return 0;
        }

        // then givens rotations to elim householder matrix
        for (int i = 0; i < j; i++) {
            T temp = H[i * n_iter + j];
            H[n_iter * i + j] = cs[i] * H[n_iter * i + j] + ss[i] * H[n_iter * (i+1) + j];
            H[n_iter * (i+1) + j] = -ss[i] * temp + cs[i] * H[n_iter * (i+1) + j];
        }

        T Hjj = H[n_iter * j + j], Hj1j = H[n_iter * (j+1) + j];
        cs[j] = Hjj / sqrt(Hjj * Hjj + Hj1j * Hj1j);
        ss[j] = cs[j] * Hj1j / Hjj;

        T g_temp = g[j];
        g[j] *= cs[j];
        g[j+1] = -ss[j] * g_temp;

        // printf("GMRES iter %d : resid %.4e\n", j, nrm_w);
        printf("GMRES iter %d : resid %.4e\n", j, abs(g[j+1]));

        if (debug) printf("j=%d, g[j]=%.4e, g[j+1]=%.4e\n", j, g[j], g[j+1]);

        H[n_iter * j + j] = cs[j] * H[n_iter * j + j] + ss[j] * H[n_iter * (j+1) + j];
        H[n_iter * (j+1) + j] = 0.0;

        if (abs(g[j+1]) < (abs_tol + beta * rel_tol)) {
            printf("GMRES converged in %d iterations to %.4e resid\n", j+1, g[j+1]);
            jj = j;
            break;
        }

        // TODO : should I use givens rotations or nrm_w for convergence? I think givens rotations
        // if (abs(nrm_w) < (abs_tol + beta * rel_tol)) {
        //     printf("GMRES converged in %d iterations to %.4e resid\n", j+1, nrm_w);
        //     jj = j;
        //     break;
        // }

    }

    // now solve Householder triangular system
    // only up to size jj+1 x jj+1 where we exited on iteration jj
    T *Hred = new T[(jj+1) * (jj+1)];
    for (int i = 0; i < jj+1; i++) {
        for (int j = 0; j < jj+1; j++) {
            // in-place transpose to be compatible with column-major hipblasDtrsv later on
            Hred[(jj+1) * i + j] = H[n_iter * j + i];

            // Hred[(jj+1) * i + j] = H[n_iter * i + j];
        }
    }

    // now print out Hred
    if (debug) {
        printf("Hred:");
        printVec<T>((jj+1) * (jj+1), Hred);
        printf("gred:");
        printVec<T>((jj+1), g);
    }

    // now copy data from Hred host to device
    T *d_Hred;
    CHECK_CUDA(hipMalloc(&d_Hred, (jj+1) * (jj+1) * sizeof(T)));
    CHECK_CUDA(hipMemcpy(d_Hred, Hred, (jj+1) * (jj+1) * sizeof(T), hipMemcpyHostToDevice));

    // also create gred vector on the device
    T *d_gred;
    CHECK_CUDA(hipMalloc(&d_gred, (jj+1) * sizeof(T)));
    CHECK_CUDA(hipMemcpy(d_gred, g, (jj+1) * sizeof(T), hipMemcpyHostToDevice));

    // now solve Householder system H * y = g
    // T *d_y;
    // CHECK_CUDA(hipMalloc(&d_y, (jj+1) * sizeof(T)));
    CHECK_CUBLAS(hipblasDtrsv(cublasHandle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, 
        HIPBLAS_DIAG_NON_UNIT, jj+1, d_Hred, jj+1, d_gred, 1));
    // writes g => y inplace

    // now copy back to the host
    T *h_y = new T[jj+1];
    CHECK_CUDA(hipMemcpy(h_y, d_gred, (jj+1) * sizeof(T), hipMemcpyDeviceToHost));

    if (debug && N <= 16) {
        printf("yred:");
        printVec<T>((jj+1), h_y);
    }

    // now compute the matrix product soln = V * y one column at a time
    // zero solution (d_x is already zero)
    for (int j = 0; j < jj+1; j++) {
        a = h_y[j];
        CHECK_CUBLAS(hipblasDaxpy(cublasHandle, N, &a, &d_Vmat[j * N], 1, d_x, 1));
    }

    // TODO : now compute the residual again
    // resid = b - A * x
    // resid = -1 * A * vj + 1 * w
    // T float_neg_one = -1.0;
    // CHECK_CUSPARSE(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &float_neg_one, matA,
    //     vec_rhs, &floatone, vec_tmp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
    //     d_bufferMV));

    // now copy solution back to host
    CHECK_CUDA(hipMemcpy(x, d_x, N * sizeof(T), hipMemcpyDeviceToHost));

    // now print solution
    if (N <= 16) {
        printf("GMRES no precond soln:");
        printVec<T>(N, x);
    }

    if (N == 16) {
        // compare against truth from python solver
        T ref[N] = {0.45454545, 0.59469697, 0.59469697, 0.45454545, 0.22348485,
            0.32954545, 0.32954545, 0.22348485, 0.10984848, 0.17045455,
            0.17045455, 0.10984848, 0.04545455, 0.0719697 , 0.0719697 ,
            0.04545455};
        T abs_diff[N], tot_abs_diff = 0.0;
        for (int i = 0; i < N; i++) {
            abs_diff[i] = abs(ref[i] - x[i]);
            tot_abs_diff += abs_diff[i];
        }
        printf("tot diff against truth = %.4e\n", tot_abs_diff);
    }

};