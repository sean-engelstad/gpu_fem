#include "hip/hip_runtime.h"
#include "_gmres_utils.h"
#include "_mat_utils.h"
#include <chrono>

int main() {
    using T = double;

    // case inputs
    // -----------

    constexpr bool test_mult = false;
    int N = 16384; // 16384
    int n_iter = min(N, 200);
    constexpr bool use_precond = true;
    constexpr bool bsr_nz_pattern = true; // default is False
    constexpr bool debug = false;
    int checkpoint = -1; // -1
    T abs_tol = 1e-8, rel_tol = 1e-8;

    // initialize data
    // ---------------
    
    int *_rowp, *_cols;
    // int M = N;
    T *_vals, *rhs, *x;
    int _nz = 5 * N - 4 * (int)sqrt((double)N);

    // allocate rowp, cols on host
    _rowp = (int*)malloc(sizeof(int) * (N + 1));
    _cols = (int*)malloc(sizeof(int) * _nz);
    _vals = (T*)malloc(sizeof(T) * _nz);
    x = (T*)malloc(sizeof(T) * N);
    rhs = (T*)malloc(sizeof(T) * N);

    for (int i = 0; i < N; i++) {
        x[i] = 0.0;
        rhs[i] = 0.0;    
    }

    // initialize data
    genLaplaceCSR<T>(_rowp, _cols, _vals, N, _nz, rhs);

    int *rowp, *cols, nz;
    T *vals;
    if constexpr (bsr_nz_pattern) {
        // convert CSR to BSR then back to CSR (so has same nz pattern)
        int *bsr_rowp, *bsr_cols, nnzb, block_dim = 2;
        T *bsr_vals;
        CSRtoBSR<T>(block_dim, N, _rowp, _cols, _vals, &bsr_rowp, &bsr_cols, &bsr_vals, &nnzb);

        // printf("nnzb %d\n", nnzb);
        // printf("bsr_rowp:");
        // printVec<int>(N/2+1, bsr_rowp);
        // printf("bsr_cols:");
        // printVec<int>(nnzb, bsr_cols);
        // printf("bsr_vals:");
        // printVec<T>(4 * nnzb, bsr_vals);

        BSRtoCSR<T>(block_dim, N, nnzb, bsr_rowp, bsr_cols, bsr_vals, &rowp, &cols, &vals, &nz);
    } else {
        rowp = _rowp;
        cols = _cols;
        vals = _vals;
        nz = _nz;
    }
    // now rhs is not zero

    // transfer data to the device
    int *d_rowp, *d_cols;
    T *d_vals, *d_x, *d_rhs, *d_vals_ILU0;
    CHECK_CUDA(hipMalloc((void **)&d_rowp, (N+1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_cols, nz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_vals, nz * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_vals_ILU0, nz * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_x, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_rhs, N * sizeof(T)));

    // copy data for the matrix over to device
    CHECK_CUDA(hipMemcpy(d_rowp, rowp, (N+1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_cols, cols, nz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_vals, vals, nz * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, N * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_rhs, rhs, N * sizeof(T), hipMemcpyHostToDevice));

    // create temp vec objects
    // -----------------------

    double *d_resid, *d_tmp, *d_w;
    CHECK_CUDA(hipMalloc((void **)&d_resid, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_tmp, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_w, N * sizeof(T)));

    // create initial cusparse and cublas objects
    // ------------------------------------------

    /* Create CUBLAS context */
    hipblasHandle_t cublasHandle = NULL;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    /* Create CUSPARSE context */
    hipsparseHandle_t cusparseHandle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    /* Description of the A matrix */
    hipsparseMatDescr_t descr = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    // wrap dense vectors into cusparse dense vector objects
    // -----------------------------------------------------

    hipsparseDnVecDescr_t vec_rhs, vec_tmp, vec_w;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_rhs, N, d_rhs, HIP_R_64F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_tmp, N, d_tmp, HIP_R_64F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_w, N, d_w, HIP_R_64F));
    
    // create the matrix CSR objects
    // -----------------------------

    hipsparseSpMatDescr_t matA = NULL;
    hipsparseSpMatDescr_t matM_lower, matM_upper;
    hipsparseFillMode_t   fill_lower    = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t   diag_unit     = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t   fill_upper    = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t   diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, N, N, nz, d_rowp, d_cols, d_vals, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    /* Copy A data to ILU(0) vals as input*/
    CHECK_CUDA(hipMemcpy(
        d_vals_ILU0, d_vals, nz*sizeof(T), hipMemcpyDeviceToDevice));
    
    //Lower Part 
    CHECK_CUSPARSE( hipsparseCreateCsr(&matM_lower, N, N, nz, d_rowp, d_cols, d_vals_ILU0,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) );

                                      CHECK_CUSPARSE( hipsparseSpMatSetAttribute(matM_lower,
        HIPSPARSE_SPMAT_FILL_MODE,
        &fill_lower, sizeof(fill_lower)) );
        CHECK_CUSPARSE( hipsparseSpMatSetAttribute(matM_lower,
        HIPSPARSE_SPMAT_DIAG_TYPE,
        &diag_unit, sizeof(diag_unit)) );

    // M_upper
    CHECK_CUSPARSE( hipsparseCreateCsr(&matM_upper, N, N, nz, d_rowp, d_cols, d_vals_ILU0,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) );
                                      CHECK_CUSPARSE( hipsparseSpMatSetAttribute(matM_upper,
            HIPSPARSE_SPMAT_FILL_MODE,
            &fill_upper, sizeof(fill_upper)) );
            CHECK_CUSPARSE( hipsparseSpMatSetAttribute(matM_upper,
            HIPSPARSE_SPMAT_DIAG_TYPE,
            &diag_non_unit,
            sizeof(diag_non_unit)) );

    // create ILU(0) preconditioner
    // ----------------------------


    int                 bufferSizeLU = 0;
    size_t              bufferSizeMV, bufferSizeL, bufferSizeU;
    void*               d_bufferLU, *d_bufferMV,  *d_bufferL, *d_bufferU;
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseMatDescr_t   matLU;
    csrilu02Info_t      infoILU = NULL;
    const T floatone = 1.0;
    const T floatzero = 0.0;

    CHECK_CUSPARSE(hipsparseCreateCsrilu02Info(&infoILU));
    CHECK_CUSPARSE( hipsparseCreateMatDescr(&matLU) );
    CHECK_CUSPARSE( hipsparseSetMatType(matLU, HIPSPARSE_MATRIX_TYPE_GENERAL) );
    CHECK_CUSPARSE( hipsparseSetMatIndexBase(matLU, HIPSPARSE_INDEX_BASE_ZERO) );

    /* Allocate workspace for cuSPARSE */
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA,
        vec_tmp, &floatzero, vec_w, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
        &bufferSizeMV));
        CHECK_CUDA( hipMalloc(&d_bufferMV, bufferSizeMV) );

    CHECK_CUSPARSE(hipsparseDcsrilu02_bufferSize(
    cusparseHandle, N, nz, matLU, d_vals_ILU0, d_rowp, d_cols, infoILU, &bufferSizeLU));
    CHECK_CUDA( hipMalloc(&d_bufferLU, bufferSizeLU) );

    CHECK_CUSPARSE( hipsparseSpSV_createDescr(&spsvDescrL) );
    CHECK_CUSPARSE(hipsparseSpSV_bufferSize(
    cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matM_lower, vec_tmp, vec_w, HIP_R_64F,
    HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL));
    CHECK_CUDA( hipMalloc(&d_bufferL, bufferSizeL) );

    CHECK_CUSPARSE( hipsparseSpSV_createDescr(&spsvDescrU) );
    CHECK_CUSPARSE( hipsparseSpSV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matM_upper, vec_tmp, vec_w, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &bufferSizeU));
    CHECK_CUDA( hipMalloc(&d_bufferU, bufferSizeU) );

    // first test the matrix-vec product on GPU
    // ----------------------------------------
    
    if constexpr (test_mult) {
        assert(4 == N);
        T *my_vec = new T[4];
        for (int i = 0; i < 4; i++) {
            my_vec[i] = i+1;
        }
        T *d_my_vec;
        CHECK_CUDA(hipMalloc((void **)&d_my_vec, 4 * sizeof(T)));
        CHECK_CUDA(hipMemcpy(d_my_vec, my_vec, 4 * sizeof(T), hipMemcpyHostToDevice));
        hipsparseDnVecDescr_t vec_my_vec = NULL;
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_my_vec, N, d_my_vec, HIP_R_64F));

        // A * my_vec => tmp1
        CHECK_CUSPARSE(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA,
            vec_my_vec, &floatzero, vec_tmp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
            d_bufferMV));

        // should get this result in tmp1
        // t1=array([  1,  -3,  -7, -11])

        // copy data back to host x vec (just a test of matmult here)
        CHECK_CUDA(hipMemcpy(x, d_tmp, N * sizeof(T), hipMemcpyDeviceToHost));
        if (debug && N == 4) {
            printf("A*my_vec=");
            printVec<T>(4, x);
        }
    }


    // GMRES solve now with CSR matrix
    // -------------------------------

    printf("checkpt\n");

    // initialize GMRES data, some on host, some on GPU
    // host GMRES data
    T g[n_iter+1], cs[n_iter], ss[n_iter];
    T H[(n_iter+1)*(n_iter)];

    // GMRES device data
    T *d_Vmat, *d_V;
    CHECK_CUDA(hipMalloc((void **)&d_Vmat, (n_iter+1) * N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_V, N * sizeof(T)));
    // use single hipsparseDnVecDescr_t of size N and just update it's values occasionally
    hipsparseDnVecDescr_t vec_V;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_V, N, d_V, HIP_R_64F));
    // other strategy is to make pointer array of hipsparseDnVecDescr_t vecs
    // update with void *col_ptr = static_cast<void*>(&d_Vmat[k * N]);
    //             hipsparseDnVecSetValues(vec_V, col_ptr)

    // setup the ILU(0) preconditioner (if in use)
    // -------------------------------------------

    if constexpr (use_precond) {
        /* Perform analysis for ILU(0) */
        CHECK_CUSPARSE(hipsparseDcsrilu02_analysis(
            cusparseHandle, N, nz, descr, d_vals_ILU0, d_rowp, d_cols, infoILU,
            HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU));

        int structural_zero;
        CHECK_CUSPARSE(hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoILU, &structural_zero));
        // print or assert if needed
        printf("structural zero = %d\n", structural_zero);

        /* generate the ILU(0) factors */
        CHECK_CUSPARSE(hipsparseDcsrilu02(
            cusparseHandle, N, nz, matLU, d_vals_ILU0, d_rowp, d_cols, infoILU,
            HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU));

        int numerical_zero;
        CHECK_CUSPARSE(hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoILU, &numerical_zero));
        // again, print/check these for zero pivots
        printf("numerical_zero = %d\n", numerical_zero);

        /* perform triangular solve analysis */
        CHECK_CUDA(hipDeviceSynchronize());
        auto start_triang = std::chrono::high_resolution_clock::now();

        CHECK_CUSPARSE(hipsparseSpSV_analysis(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone,
            matM_lower, vec_tmp, vec_w, HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, d_bufferL));
        CHECK_CUSPARSE(hipsparseSpSV_analysis(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone,
            matM_upper, vec_tmp, vec_w, HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, d_bufferU));

        CHECK_CUDA(hipDeviceSynchronize());
        auto end_triang = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> triang_time_loc = end_triang - start_triang;
        printf("Double CSR triang solve:");
        printf("\ttriang solve time %.4e\n", triang_time_loc.count());

    }

    if constexpr (debug && use_precond) {
        // print A matrix values::
        T *h_A = new T[nz];
        T *h_M = new T[nz];
        CHECK_CUDA(hipMemcpy(h_A, d_vals, nz * sizeof(T), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_M, d_vals_ILU0, nz * sizeof(T), hipMemcpyDeviceToHost));
        
        printf("h_A:");
        printVec<T>(nz, h_A);
        printf("h_M:");
        printVec<T>(nz, h_M);

        // test LU solve on each unit vector..
    }


    // GMRES algorithm
    // ----------------------------

    int jj = n_iter - 1;

    // apply precond to rhs if in use
    if constexpr (use_precond) {
        // print part of initial vec_rhs
        int NPRINT = N;
        T *h_rhs = new T[NPRINT];
        CHECK_CUDA(hipMemcpy(h_rhs, d_rhs, NPRINT * sizeof(T), hipMemcpyDeviceToHost));
        if (debug) {
            printf("b:");
            printVec<T>(NPRINT, h_rhs);
        }

        // zero vec_tmp
        CHECK_CUDA(hipMemset(d_tmp, 0.0, N * sizeof(T)));
        
        // preconditioner application: d_zm1 = U^-1 L^-1 d_r
        CHECK_CUSPARSE(hipsparseSpSV_solve(cusparseHandle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone,
            matM_lower, vec_rhs, vec_tmp, HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT,
            spsvDescrL) );

        if (debug) {
            CHECK_CUDA(hipMemcpy(h_rhs, d_tmp, NPRINT * sizeof(T), hipMemcpyDeviceToHost));
            printf("L^-1 * b:");
            printVec<T>(NPRINT, h_rhs);
        }
            
        CHECK_CUSPARSE(hipsparseSpSV_solve(cusparseHandle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matM_upper,
            vec_tmp, vec_rhs,
            HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT,
            spsvDescrU));

        if (debug) {
            CHECK_CUDA(hipMemcpy(h_rhs, d_rhs, NPRINT * sizeof(T), hipMemcpyDeviceToHost));
            printf("U^-1 * L^-1 * b:");
            printVec<T>(NPRINT, h_rhs);
        }
    }

    // GMRES initial residual
    // assumes here d_X is 0 initially => so r0 = b - Ax = b
    T beta;
    CHECK_CUBLAS(hipblasDnrm2(cublasHandle, N, d_rhs, 1, &beta));
    printf("GMRES init resid = %.9e\n", beta);
    g[0] = beta;

    if (debug && checkpoint == 1) return 0;

    // set v0 = r0 / beta (unit vec)
    T a = 1.0 / beta;
    CHECK_CUBLAS(hipblasDaxpy(cublasHandle, N, &a, d_rhs, 1, &d_Vmat[0], 1));

    T *h_v0 = new T[N];
    CHECK_CUDA(hipMemcpy(h_v0, d_Vmat, N * sizeof(T), hipMemcpyDeviceToHost));
    // print vec
    if (debug) {
        printf("r0:");
        printVec<T>(4, h_v0);
    }

    // then begin main GMRES iteration loop!
    for (int j = 0; j < n_iter; j++) {
        // zero this vec
        // CHECK_CUDA(hipMemset(&d_Vmat[j * N], 0.0, N * sizeof(T)));

        // get vj and copy it into the cusparseDnVec_t
        void *vj_col = static_cast<void*>(&d_Vmat[j * N]);
        CHECK_CUSPARSE(hipsparseDnVecSetValues(vec_V, vj_col));

        // w = A * vj + 0 * w
        CHECK_CUSPARSE(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA,
            vec_V, &floatzero, vec_w, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
            d_bufferMV));

        if (debug && N <= 16) {
            T *h_w = new T[N];
            // printf("checkpt3\n");
            CHECK_CUDA(hipMemcpy(h_w, d_w, N * sizeof(T), hipMemcpyDeviceToHost));
            printf("w=A*v:");
            printVec<T>(N, h_w);

            // if (j == 0) return 0;
        }

        if constexpr (use_precond) {
            // preconditioner application: d_zm1 = U^-1 L^-1 d_r
            CHECK_CUSPARSE(hipsparseSpSV_solve(cusparseHandle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone,
                matM_lower, vec_w, vec_tmp, HIP_R_64F,
                HIPSPARSE_SPSV_ALG_DEFAULT,
                spsvDescrL) );
                
            CHECK_CUSPARSE(hipsparseSpSV_solve(cusparseHandle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matM_upper,
                vec_tmp, vec_w,
                HIP_R_64F,
                HIPSPARSE_SPSV_ALG_DEFAULT,
                spsvDescrU));
        }

        // double check and print the value of 
        if (debug && N <= 16) {
            T *h_w = new T[N];
            // printf("checkpt3\n");
            CHECK_CUDA(hipMemcpy(h_w, d_w, N * sizeof(T), hipMemcpyDeviceToHost));
            printf("h_w[%d] pre-GS:", j);
            printVec<T>(N, h_w);

            // if (j == 0) return 0;
        }

        if (debug && checkpoint == 2) return 0;

        // now update householder matrix
        for (int i = 0 ; i < j+1; i++) {
            // get vi column
            void *vi_col = static_cast<void*>(&d_Vmat[i * N]);
            CHECK_CUSPARSE(hipsparseDnVecSetValues(vec_V, vi_col));

            T w_vi_dot;
            CHECK_CUBLAS(hipblasDdot(cublasHandle, N, d_w, 1, &d_Vmat[i * N], 1, &w_vi_dot));

            // H_ij = vi dot w
            H[n_iter * i + j] = w_vi_dot;

            if (debug) printf("H[%d,%d] = %.9e\n", i, j, H[n_iter * i + j]);
            
            // w -= Hij * vi
            a = -H[n_iter * i + j];
            CHECK_CUBLAS(hipblasDaxpy(cublasHandle, N, &a, &d_Vmat[i * N], 1, d_w, 1));
        }

        // double check and print the value of 
        if (debug && N <= 16) {
            T *h_w = new T[N];
            // printf("checkpt3\n");
            CHECK_CUDA(hipMemcpy(h_w, d_w, N * sizeof(T), hipMemcpyDeviceToHost));
            printf("h_w[%d] post GS:", j);
            printVec<T>(N, h_w);

            // if (j == 0) return 0;
        }

        if (debug && checkpoint == 3) return 0;

        // norm of w
        T nrm_w;
        CHECK_CUBLAS(hipblasDnrm2(cublasHandle, N, d_w, 1, &nrm_w));

        // H_{j+1,j}
        H[n_iter * (j+1) + j] = nrm_w;

        // v_{j+1} column unit vec = w / H_{j+1,j}
        a = 1.0 / H[n_iter * (j+1) + j];
        CHECK_CUBLAS(hipblasDcopy(cublasHandle, N, d_w, 1, &d_Vmat[(j+1) * N], 1));
        CHECK_CUBLAS(hipblasDscal(cublasHandle, N, &a, &d_Vmat[(j+1) * N], 1));

        if (debug && N <= 16) {
            T *h_tmp = new T[N];
            // printf("checkpt3\n");
            CHECK_CUDA(hipMemcpy(h_tmp, &d_Vmat[(j+1) * N], N * sizeof(T), hipMemcpyDeviceToHost));
            printf("next V:");
            printVec<T>(N, h_tmp);

            // if (j == 0) return 0;
        }

        // then givens rotations to elim householder matrix
        for (int i = 0; i < j; i++) {
            T temp = H[i * n_iter + j];
            H[n_iter * i + j] = cs[i] * H[n_iter * i + j] + ss[i] * H[n_iter * (i+1) + j];
            H[n_iter * (i+1) + j] = -ss[i] * temp + cs[i] * H[n_iter * (i+1) + j];
        }

        T Hjj = H[n_iter * j + j], Hj1j = H[n_iter * (j+1) + j];
        cs[j] = Hjj / sqrt(Hjj * Hjj + Hj1j * Hj1j);
        ss[j] = cs[j] * Hj1j / Hjj;

        T g_temp = g[j];
        g[j] *= cs[j];
        g[j+1] = -ss[j] * g_temp;

        // printf("GMRES iter %d : resid %.9e\n", j, nrm_w);
        printf("GMRES iter %d : resid %.9e\n", j, abs(g[j+1]));

        if (debug) printf("j=%d, g[j]=%.9e, g[j+1]=%.9e\n", j, g[j], g[j+1]);

        H[n_iter * j + j] = cs[j] * H[n_iter * j + j] + ss[j] * H[n_iter * (j+1) + j];
        H[n_iter * (j+1) + j] = 0.0;

        if (abs(g[j+1]) < (abs_tol + beta * rel_tol)) {
            printf("GMRES converged in %d iterations to %.9e resid\n", j+1, g[j+1]);
            jj = j;
            break;
        }

        // TODO : should I use givens rotations or nrm_w for convergence? I think givens rotations
        // if (abs(nrm_w) < (abs_tol + beta * rel_tol)) {
        //     printf("GMRES converged in %d iterations to %.9e resid\n", j+1, nrm_w);
        //     jj = j;
        //     break;
        // }

    }

    // now solve Householder triangular system
    // only up to size jj+1 x jj+1 where we exited on iteration jj
    T *Hred = new T[(jj+1) * (jj+1)];
    for (int i = 0; i < jj+1; i++) {
        for (int j = 0; j < jj+1; j++) {
            // in-place transpose to be compatible with column-major hipblasDtrsv later on
            Hred[(jj+1) * i + j] = H[n_iter * j + i];

            // Hred[(jj+1) * i + j] = H[n_iter * i + j];
        }
    }

    // now print out Hred
    if (debug) {
        printf("Hred:");
        printVec<T>((jj+1) * (jj+1), Hred);
        printf("gred:");
        printVec<T>((jj+1), g);
    }

    // now copy data from Hred host to device
    T *d_Hred;
    CHECK_CUDA(hipMalloc(&d_Hred, (jj+1) * (jj+1) * sizeof(T)));
    CHECK_CUDA(hipMemcpy(d_Hred, Hred, (jj+1) * (jj+1) * sizeof(T), hipMemcpyHostToDevice));

    // also create gred vector on the device
    T *d_gred;
    CHECK_CUDA(hipMalloc(&d_gred, (jj+1) * sizeof(T)));
    CHECK_CUDA(hipMemcpy(d_gred, g, (jj+1) * sizeof(T), hipMemcpyHostToDevice));

    // now solve Householder system H * y = g
    // T *d_y;
    // CHECK_CUDA(hipMalloc(&d_y, (jj+1) * sizeof(T)));
    CHECK_CUBLAS(hipblasDtrsv(cublasHandle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, 
        HIPBLAS_DIAG_NON_UNIT, jj+1, d_Hred, jj+1, d_gred, 1));
    // writes g => y inplace

    // now copy back to the host
    T *h_y = new T[jj+1];
    CHECK_CUDA(hipMemcpy(h_y, d_gred, (jj+1) * sizeof(T), hipMemcpyDeviceToHost));

    if (debug && N <= 16) {
        printf("yred:");
        printVec<T>((jj+1), h_y);
    }

    // now compute the matrix product soln = V * y one column at a time
    // zero solution (d_x is already zero)
    for (int j = 0; j < jj+1; j++) {
        a = h_y[j];
        CHECK_CUBLAS(hipblasDaxpy(cublasHandle, N, &a, &d_Vmat[j * N], 1, d_x, 1));
    }

    // now compute the residual again
    // resid = b - A * x
    // resid = -1 * A * vj + 1 * w
    T float_neg_one = -1.0;
    CHECK_CUSPARSE(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &float_neg_one, matA,
        vec_rhs, &floatone, vec_tmp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
        d_bufferMV));

    // now copy solution back to host
    CHECK_CUDA(hipMemcpy(x, d_x, N * sizeof(T), hipMemcpyDeviceToHost));

    // now print solution
    if (N <= 16) {
        printf("GMRES no precond soln:");
        printVec<T>(N, x);
    }

    if (N == 16) {
        // compare against truth from python solver
        T ref[N] = {0.45454545, 0.59469697, 0.59469697, 0.45454545, 0.22348485,
            0.32954545, 0.32954545, 0.22348485, 0.10984848, 0.17045455,
            0.17045455, 0.10984848, 0.04545455, 0.0719697 , 0.0719697 ,
            0.04545455};
        T abs_diff[N], tot_abs_diff = 0.0;
        for (int i = 0; i < N; i++) {
            abs_diff[i] = abs(ref[i] - x[i]);
            tot_abs_diff += abs_diff[i];
        }
        printf("tot diff against truth = %.9e\n", tot_abs_diff);
    }

};