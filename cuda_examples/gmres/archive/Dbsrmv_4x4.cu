#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define CHECK_CUDA(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

#define CHECK_CUSPARSE(call)                                     \
    {                                                            \
        hipsparseStatus_t err = call;                             \
        if (err != HIPSPARSE_STATUS_SUCCESS) {                    \
            std::cerr << "CUSPARSE error: " << err << std::endl; \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

int main() {
    using T = double;

    // problem
    // A = [1 * e1; 2 * e2; 3 * e3; 4 * e4]
    // x = [1, 3, 4, 1]
    // y = A * x = [1, 6, 12, 4]

    // initialize host data
    int rowp[3] = {0, 1, 2};
    int cols[2] = {0, 1};
    T vals[8] = {1.0, 0.0, 0.0, 2.0, 3.0, 0.0, 0.0, 4.0};
    T x[4] = {1, 3, 4, 1};
    T *y = new T[4];

    int *d_rowp, *d_cols;
    T *d_vals, *d_x, *d_y;
    CHECK_CUDA(hipMalloc((void **)&d_rowp, 3 * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_cols, 2 * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_vals, 8 * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_x, 4 * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_y, 4 * sizeof(T)));

    CHECK_CUDA(hipMemcpy(d_rowp, rowp, 3 * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_cols, cols, 2 * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_vals, vals, 8 * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, 4 * sizeof(T), hipMemcpyHostToDevice));

    /* Description of the A matrix */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    /* Create CUSPARSE context */
    hipsparseHandle_t cusparseHandle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    int mb = 2;
    int block_dim = 2;
    T a = 1.0, b = 0.0;
    int nnzb = 2;

    CHECK_CUSPARSE(hipsparseDbsrmv(
        cusparseHandle, 
        HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        mb, mb, nnzb,
        &a, descrA,
        d_vals, d_rowp, d_cols,
        block_dim,
        d_x,
        &b,
        d_y
    ));

    // copy d_y back to host y vec
    // T y_ref[2] = {1, 6, 12, 4};
    CHECK_CUDA(hipMemcpy(y, d_y, 4 * sizeof(T), hipMemcpyDeviceToHost));
    printf("y:");
    for (int i = 0; i < 4; i++) {
        printf("%.4e,", y[i]);
    }
    printf("\n");

    // Cleanup
    CHECK_CUDA(hipFree(d_rowp));
    CHECK_CUDA(hipFree(d_cols));
    CHECK_CUDA(hipFree(d_vals));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
};