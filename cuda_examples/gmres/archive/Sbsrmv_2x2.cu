#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define CHECK_CUDA(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

#define CHECK_CUSPARSE(call)                                     \
    {                                                            \
        hipsparseStatus_t err = call;                             \
        if (err != HIPSPARSE_STATUS_SUCCESS) {                    \
            std::cerr << "CUSPARSE error: " << err << std::endl; \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

int main() {
    using T = float;

    // problem
    // A = [1, 0; 0, 2]
    // x = [3, 4]
    // y = A * x = [3, 8] (Demonstrate with BSR mv operation)

    // initialize host data
    int rowp[2] = {0, 1};
    int cols[1] = {0};
    T vals[4] = {1.0, 0.0, 0.0, 2.0};
    T x[2] = {3, 4.0};
    T *y = new T[2];

    int *d_rowp, *d_cols;
    T *d_vals, *d_x, *d_y;
    CHECK_CUDA(hipMalloc((void **)&d_rowp, 2 * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_cols, 1 * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_vals, 4 * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_x, 2 * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_y, 2 * sizeof(T)));

    CHECK_CUDA(hipMemcpy(d_rowp, rowp, 2 * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_cols, cols, 1 * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_vals, vals, 4 * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, 2 * sizeof(T), hipMemcpyHostToDevice));

    /* Description of the A matrix */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    /* Create CUSPARSE context */
    hipsparseHandle_t cusparseHandle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    int mb = 1;
    int block_dim = 2;
    T a = 1.0, b = 0.0;
    int nnzb = 1;

    CHECK_CUSPARSE(hipsparseSbsrmv(
        cusparseHandle, 
        HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        mb, mb, nnzb,
        &a, descrA,
        d_vals, d_rowp, d_cols,
        block_dim,
        d_x,
        &b,
        d_y
    ));

    // copy d_y back to host y vec
    // T y_ref[2] = {3, 8};
    CHECK_CUDA(hipMemcpy(y, d_y, 2 * sizeof(T), hipMemcpyDeviceToHost));
    printf("y:");
    for (int i = 0; i < 2; i++) {
        printf("%.4e,", y[i]);
    }
    printf("\n");

    // Cleanup
    CHECK_CUDA(hipFree(d_rowp));
    CHECK_CUDA(hipFree(d_cols));
    CHECK_CUDA(hipFree(d_vals));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
};