#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define CHECK_CUDA(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

#define CHECK_CUSPARSE(call)                                     \
    {                                                            \
        hipsparseStatus_t err = call;                             \
        if (err != HIPSPARSE_STATUS_SUCCESS) {                    \
            std::cerr << "CUSPARSE error: " << err << std::endl; \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    }

int main() {
    using T = double;

    // problem
    // A = [1, 0; 0, 2]
    // x = [3, 4]
    // y = A * x = [3, 8] (Demonstrate with BSR mv operation)

    // initialize host data
    int rowp[3] = {0, 1, 2};
    int cols[2] = {0, 1};
    T vals[2] = {1.0, 2.0};
    T x[2] = {3, 4};
    T y[2];

    int *d_rowp, *d_cols;
    T *d_vals, *d_x, *d_y;
    CHECK_CUDA(hipMalloc((void **)&d_rowp, 3 * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_cols, 2 * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_vals, 2 * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_x, 2 * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_y, 2 * sizeof(T)));

    CHECK_CUDA(hipMemcpy(d_rowp, rowp, 3 * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_cols, cols, 2 * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_vals, vals, 2 * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, 2 * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, y, 2 * sizeof(T), hipMemcpyHostToDevice));

    /* Description of the A matrix */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    /* Create CUSPARSE context */
    hipsparseHandle_t cusparseHandle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    int mb = 2;
    int block_dim = 1;
    T a = 1.0, b = 0.0;
    int nnzb = 2;

    CHECK_CUSPARSE(hipsparseDbsrmv(
        cusparseHandle, 
        HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        mb, mb, nnzb,
        &a, descrA,
        d_vals, d_rowp, d_cols,
        block_dim,
        d_x,
        &b,
        d_y
    ));

    // copy d_y back to host y vec
    T y_ref[2] = {3, 8};
    CHECK_CUDA(hipMemcpy(&y[0], d_y, 2 * sizeof(T), hipMemcpyDeviceToHost));
    printf("y:");
    for (int i = 0; i < 2; i++) {
        printf("%.4e,", y[i]);
    }
};