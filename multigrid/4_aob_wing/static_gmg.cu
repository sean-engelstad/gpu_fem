
#include "linalg/_linalg.h"
#include "mesh/TACSMeshLoader.h"
#include "mesh/vtk_writer.h"
#include "solvers/_solvers.h"

// shell imports
#include "assembler.h"
#include "element/shell/physics/isotropic_shell.h"
#include "element/shell/shell_elem_group.h"

// local multigrid imports
#include "multigrid/grid.h"
#include "multigrid/fea.h"
#include "multigrid/solvers/gmg.h"
#include <string>
#include <chrono>

/* argparse options:
[mg/direct/debug] [--level int]
*/

void to_lowercase(char *str) {
    for (; *str; ++str) {
        *str = std::tolower(*str);
    }
}

std::string time_string(int itime) {
    std::string _time = std::to_string(itime);
    if (itime < 10) {
        return "00" + _time;
    } else if (itime < 100) {
        return "0" + _time;
    } else {
        return _time;
    }
}

void solve_linear_multigrid(MPI_Comm &comm, int level, double SR, int nsmooth) {
    // geometric multigrid method here..
    // need to make a number of grids..
    // level gives the finest level here..

    using T = double;   
    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;
    constexpr bool has_ref_axis = false;
    constexpr bool is_nonlinear = false;
    using Data = ShellIsotropicData<T, has_ref_axis>;
    using Physics = IsotropicShell<T, Data, is_nonlinear>;
    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
    using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

    // old smoothers
    // const SMOOTHER smoother = LEXIGRAPHIC_GS;
    // const SMOOTHER smoother = MULTICOLOR_GS;
    // const SMOOTHER smoother = MULTICOLOR_GS_FAST;
    // const SMOOTHER smoother = MULTICOLOR_GS_FAST2; // fastest (faster than MULTICOLOR_GS_FAST by about 2.6x at high DOF)
    // const SMOOTHER smoother = DAMPED_JACOBI;
    const SMOOTHER smoother = MULTICOLOR_GS_FAST2_JUNCTION;

    const SCALER scaler = LINE_SEARCH;

    const bool is_bsr = true; // need this one if want to smooth prolongation
    // const bool is_bsr = false; // no difference in intra-nodal (default old working prolong)
    using Prolongation = UnstructuredProlongation<Basis, is_bsr>; 

    using GRID = ShellGrid<Assembler, Prolongation, smoother, scaler>;
    using MG = GeometricMultigridSolver<GRID>;

    auto start0 = std::chrono::high_resolution_clock::now();
    auto mg = MG();

    // make each wing multigrid object.. (highest mesh level is finest, this is flipped from MG object's convention)
    for (int i = level; i >= 0; i--) {

        // read the ESP/CAPS => nastran mesh for TACS
        TACSMeshLoader mesh_loader{comm};
        std::string fname = "meshes/aob_wing_L" + std::to_string(i) + ".bdf";
        mesh_loader.scanBDFFile(fname.c_str());
        double E = 70e9, nu = 0.3, thick = 2.0 / SR;  // material & thick properties (start thicker first try)
        // double E = 70e9, nu = 0.3, thick = 1.0;  // material & thick properties (start thicker first try)
        // double E = 70e9, nu = 0.3, thick = 0.01;  // material & thick properties (start thicker first try)
        // double E = 70e9, nu = 0.3, thick = 0.005;  // material & thick properties

        printf("making assembler+GMG for mesh '%s'\n", fname.c_str());
        
        // create the TACS Assembler from the mesh loader
        auto assembler = Assembler::createFromBDF(mesh_loader, Data(E, nu, thick));

        // create the loads (really only needed on finer mesh.. TBD how to setup nonlinear case..)
        int nvars = assembler.get_num_vars();
        int nnodes = assembler.get_num_nodes();
        HostVec<T> h_loads(nvars);
        double load_mag = 10.0;
        double *my_loads = h_loads.getPtr();
        for (int inode = 0; inode < nnodes; inode++) {
            my_loads[6 * inode + 2] = load_mag;
        }

        // TODO : run optimized design from AOB case

        // make the grid
        bool full_LU = i == 0; // smallest grid is direct solve
        bool reorder;
        if (smoother == LEXIGRAPHIC_GS) {
            reorder = false;
        } else if (smoother == MULTICOLOR_GS || smoother == MULTICOLOR_GS_FAST || smoother == MULTICOLOR_GS_FAST2 
            || smoother == MULTICOLOR_GS_FAST2_JUNCTION) {
            reorder = true;
        } else if (smoother == DAMPED_JACOBI) {
            reorder = false;
        }
        // printf("reorder %d\n", reorder);
        auto grid = *GRID::buildFromAssembler(assembler, my_loads, full_LU, reorder);
        mg.grids.push_back(grid); // add new grid
    }

    if (!Prolongation::structured) {
        printf("begin unstructured map\n");
        // int ELEM_MAX = 4; // for plate, cylinder
        int ELEM_MAX = 10; // for wingbox esp near rib, spar, OML junctions
        mg.template init_unstructured<Basis>(ELEM_MAX);
        printf("done with init unstructured\n");
        // return; // TEMP DEBUG
    }

    auto end0 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> startup_time = end0 - start0;

    T init_resid_nrm = mg.grids[0].getResidNorm();

    CHECK_CUDA(hipDeviceSynchronize());
    auto start1 = std::chrono::high_resolution_clock::now();
    printf("starting v cycle solve\n");
    int pre_smooth = nsmooth, post_smooth = nsmooth;
    // best was V(4,4) before
    // bool print = false;
    bool print = true;
    T atol = 1e-6, rtol = 1e-6;
    T omega = 1.5; // good GS-SSOR parameter (speedups up convergence)
    // T omega = 1.0;
    // T omega = 0.85;
    if (smoother == LEXIGRAPHIC_GS) omega = 1.4;
    if (smoother == DAMPED_JACOBI) omega = 0.7; // damped jacobi diverges on wingbox
    int n_cycles = 200;
    if (SR > 100.0) n_cycles = 1000;

    bool time = false;
    // bool time = true;

    bool symmetric = false;
    // bool symmetric = true;

    // int print_freq = 1;
    int print_freq = 5;

    // bool double_smooth = false;
    bool double_smooth = true; // false
    mg.vcycle_solve(0, pre_smooth, post_smooth, n_cycles, print, atol, rtol, omega, double_smooth, print_freq, symmetric, time);
    // mg.wcycle_solve(0, pre_smooth, post_smooth, n_cycles, print, atol, rtol, omega);
    
    CHECK_CUDA(hipDeviceSynchronize());
    auto end1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> solve_time = end1 - start1;
    int ndof = mg.grids[0].N;
    double total = startup_time.count() + solve_time.count();
    double mem_MB = mg.get_memory_usage_mb();
    printf("wingbox GMG solve, ndof %d : startup time %.2e, solve time %.2e, total %.2e, with mem(MB) %.2e\n", ndof, startup_time.count(), solve_time.count(), total, mem_MB);

    // double check with true resid nrm
    T resid_nrm = mg.grids[0].getResidNorm();
    printf("init resid_nrm = %.2e => final resid_nrm = %.2e\n", init_resid_nrm, resid_nrm);

    // print some of the data of host residual
    int *d_perm = mg.grids[0].d_perm;
    auto h_soln = mg.grids[0].d_soln.createPermuteVec(6, d_perm).createHostVec();
    printToVTK<Assembler,HostVec<T>>(mg.grids[0].assembler, h_soln, "out/aob_wing_mg.vtk");
}

void solve_linear_direct(MPI_Comm &comm, int level, double SR) {
  using T = double;

  auto start0 = std::chrono::high_resolution_clock::now();

  TACSMeshLoader mesh_loader{comm};
  std::string fname = "meshes/aob_wing_L" + std::to_string(level) + ".bdf";
  mesh_loader.scanBDFFile(fname.c_str());

  using Quad = QuadLinearQuadrature<T>;
  using Director = LinearizedRotation<T>;
  using Basis = ShellQuadBasis<T, Quad, 2>;
  using Geo = Basis::Geo;

  constexpr bool has_ref_axis = false;
  constexpr bool is_nonlinear = false;
  using Data = ShellIsotropicData<T, has_ref_axis>;
  using Physics = IsotropicShell<T, Data, is_nonlinear>;

  using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
  using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;
  using GRID = ShellGrid<Assembler, UnstructuredProlongation<Basis,true>, MULTICOLOR_GS_FAST2_JUNCTION, NONE>;

  //   double E = 70e9, nu = 0.3, thick = 0.005;  // material & thick properties
  double E = 70e9, nu = 0.3, thick = 2.0 / SR;  // material & thick properties

  // make the assembler from the uCRM mesh
  auto assembler = Assembler::createFromBDF(mesh_loader, Data(E, nu, thick));

  // TODO : set this in from optimized design from AOB case

  // BSR factorization
  auto& bsr_data = assembler.getBsrData();
  double fillin = 10.0;  // 10.0
  bool print = true;
  bsr_data.AMD_reordering();

//   // TRY INSTEAD Mc REORDERING
//   int num_colors, *_color_rowp, *nodal_num_comps, *node_geom_ind;
//   GRID::get_nodal_geom_indices(assembler, nodal_num_comps, node_geom_ind);
//   bsr_data.multicolor_junction_reordering_v2(node_geom_ind, num_colors, _color_rowp);

  bsr_data.compute_full_LU_pattern(fillin, print);
  assembler.moveBsrDataToDevice();

  // get the loads
  int nvars = assembler.get_num_vars();
  int nnodes = assembler.get_num_nodes();
  HostVec<T> h_loads(nvars);
  double load_mag = 10.0;
  double *h_loads_ptr = h_loads.getPtr();
  for (int inode = 0; inode < nnodes; inode++) {
    h_loads_ptr[6 * inode + 2] = load_mag;
  }
  auto loads = h_loads.createDeviceVec();
  assembler.apply_bcs(loads);

  // setup kmat and initial vecs
  auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);
  auto soln = assembler.createVarsVec();
  auto res = assembler.createVarsVec();
  auto vars = assembler.createVarsVec();

  // assemble the kmat
  assembler.set_variables(vars);
  assembler.add_jacobian(res, kmat);
  assembler.apply_bcs(res);
  assembler.apply_bcs(kmat);

  CHECK_CUDA(hipDeviceSynchronize());
  auto start1 = std::chrono::high_resolution_clock::now();

  // solve the linear system
  CUSPARSE::direct_LU_solve(kmat, loads, soln);

  CHECK_CUDA(hipDeviceSynchronize());
  auto end1 = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> solve_time = end1 - start1;

  size_t bytes_per_double = sizeof(double);
  double mem_mb = static_cast<double>(bytes_per_double) * static_cast<double>(bsr_data.nnzb) * 36.0 / 1024.0 / 1024.0;
  printf("direct LU solve uses memory(MB) %.2e in %.2e sec\n", mem_mb, solve_time.count());

  // print some of the data of host residual
  auto h_soln = soln.createHostVec();
  printToVTK<Assembler, HostVec<T>>(assembler, h_soln, "out/aob_direct_L" + std::to_string(level) + ".vtk");

  // free data
  assembler.free();
  h_loads.free();
  kmat.free();
  soln.free();
  res.free();
  vars.free();
  h_soln.free();
}

int main(int argc, char **argv) {

    // Intialize MPI and declare communicator
    MPI_Init(&argc, &argv);
    MPI_Comm comm = MPI_COMM_WORLD;

    // DEFAULTS
    int level = 0; // level mesh to solve..
    bool is_multigrid = true;
    // bool is_debug = false;
    double SR = 50.0;
    // int nsmooth = 4;
    int nsmooth = 6; // typically faster right now

    // Parse arguments
    for (int i = 1; i < argc; ++i) {
        char* arg = argv[i];
        to_lowercase(arg);

        if (strcmp(arg, "direct") == 0) {
            is_multigrid = false;
        } else if (strcmp(arg, "mg") == 0) {
            is_multigrid = true;
        } else if (strcmp(arg, "--sr") == 0) {
            if (i + 1 < argc) {
                SR = std::atof(argv[++i]);
            } else {
                std::cerr << "Missing value for --SR\n";
                return 1;
            }
        } else if (strcmp(arg, "--level") == 0) {
            if (i + 1 < argc) {
                level = std::atoi(argv[++i]);
            } else {
                std::cerr << "Missing value for --level\n";
                return 1;
            }
        } else if (strcmp(arg, "--nsmooth") == 0) {
            if (i + 1 < argc) {
                nsmooth = std::atoi(argv[++i]);
            } else {
                std::cerr << "Missing value for --nsmooth\n";
                return 1;
            }
        } else {
            std::cerr << "Unknown argument: " << argv[i] << std::endl;
            std::cerr << "Usage: " << argv[0] << " [direct/mg] [--level int] [--SR double] [--nsmooth int]" << std::endl;
            return 1;
        }
    }

    // solve linear with directLU solve
    if (is_multigrid) { // && !is_debug) {
        solve_linear_multigrid(comm, level, SR, nsmooth);
    // } else if (is_multigrid && is_debug) {
    //     solve_linear_multigrid_debug(comm, level, SR);
    } else {
        solve_linear_direct(comm, level, SR);
    }

    // TBD multigrid solve..

    MPI_Finalize();
    return 0;
};
