// general gpu_fem imports
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"
#include "mesh/TACSMeshLoader.h"
#include "mesh/vtk_writer.h"

// shell imports
#include "assembler.h"
#include "element/shell/shell_elem_group.h"
#include "element/shell/physics/isotropic_shell.h"

// local multigrid imports
#include "multigrid/grid.h"
#include "multigrid/fea.h"
// #include "multigrid/solvers/gmg.h"
#include <string>
#include <chrono>

// new multigrid imports for K-cycles, etc.
#include "multigrid/solvers/solve_utils.h"
#include "multigrid/solvers/direct/cusp_directLU.h"
#include "multigrid/solvers/krylov/bsr_pcg.h"
#include "multigrid/solvers/multilevel/kcycle.h"
#include "multigrid/solvers/multilevel/twolevel.h"

// optimization with GMG imports
#include "multigrid/interface.h"

void to_lowercase(char *str) {
    for (; *str; ++str) {
        *str = std::tolower(*str);
    }
}

void multigrid_plate_solve(int nxe, double SR, int n_vcycles) {
    // geometric multigrid method here..
    // need to make a number of grids..

    using T = double;   
    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;
    constexpr bool has_ref_axis = false;
    constexpr bool is_nonlinear = false;
    using Data = ShellIsotropicData<T, has_ref_axis>;
    using Physics = IsotropicShell<T, Data, is_nonlinear>;
    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
    using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

    // multigrid objects
    const SMOOTHER smoother = MULTICOLOR_GS_FAST2; // this is much faster than other two methods (MULTICOLOR_GS_FAST is about 2.6x slower at high DOF)
    const SCALER scaler  = LINE_SEARCH;
    using Prolongation = StructuredProlongation<PLATE>;
    using GRID = ShellGrid<Assembler, Prolongation, smoother, scaler>;
    // using MG = ShellMultigrid<GRID>;

    using DirectSolve = CusparseMGDirectLU<GRID>;
    using KrylovSolve = PCGSolver<T, GRID>;
    using TwoLevelSolve = MultigridTwoLevelSolver<GRID>;
    using KMG = MultilevelKcycleSolver<GRID, DirectSolve, TwoLevelSolve, KrylovSolve>;
    using MGInterface = TacsMGInterface<T, Assembler, KMG>;

    
    auto mg = KMG();

    // get nxe_min for not exactly power of 2 case
    int pre_nxe_min = nxe > 32 ? 32 : 4;
    int nxe_min = pre_nxe_min;
    for (int c_nxe = nxe; c_nxe >= pre_nxe_min; c_nxe /= 2) {
        nxe_min = c_nxe;
    }
    
    // set the number of design variables (can increase later..)
    int nxe_dv = 4, nye_dv = 4;

    // make each grid
    for (int c_nxe = nxe; c_nxe >= nxe_min; c_nxe /= 2) {
        // make the assembler
        int c_nye = c_nxe;
        double Lx = 1.0, Ly = 1.0, E = 70e9, nu = 0.3, thick = 1.0 / SR, rho = 2500, ys = 350e6;
        int nxe_per_comp = c_nxe / nxe_dv, nye_per_comp = c_nye/nye_dv; 
        auto assembler = createPlateAssembler<Assembler>(c_nxe, c_nye, Lx, Ly, E, nu, thick, rho, ys, nxe_per_comp, nye_per_comp);
        double Q = 1.0; // load magnitude
        T *my_loads = getPlateLoads<T, Physics>(c_nxe, c_nye, Lx, Ly, Q);
        printf("making grid with nxe %d\n", c_nxe);

        // make the grid
        bool full_LU = c_nxe == nxe_min; // smallest grid is direct solve
        bool reorder = true; // color reorder
        auto grid = *GRID::buildFromAssembler(assembler, my_loads, full_LU, reorder);
        mg.grids.push_back(grid); // add new grid
    }

    bool print = true;
    // bool double_smooth = true;
    int nsmooth = 1, ninnercyc = 2, print_freq = 3;
    int n_krylov = 100;
    T atol = 1e-6, rtol = 1e-6;
    T omega = 0.85;
    mg.init_outer_solver(nsmooth, ninnercyc, n_krylov, omega, atol, rtol, print_freq, print);

    // now make the solver interface
    auto interface = MGInterface(mg, print);
    // T atol = 1e-6, rtol = 1e-6;
    // int n_cycles = 200, pre_smooth = 1, post_smooth = 1, print_freq = 3;
    // interface.set_mg_solver_settings(rtol, atol, n_cycles, pre_smooth, post_smooth, print_freq);

    // get struct loads on finest grid
    auto fine_grid = mg.grids[0];
    DeviceVec<T> d_loads(fine_grid.N);
    mg.grids[0].getDefect(d_loads);

    // get initial dvs
    int ndvs = mg.grids[0].assembler.get_num_dvs();
    T thick = 1.0 / SR;
    auto d_dvs = DeviceVec<T>(ndvs, thick);

    // now do a linear static solve with GMG
    interface.solve(d_loads);
    interface.writeSoln("out/plate_mg1.vtk");

    // define function objects
    auto mass = Mass<T, DeviceVec>();
    T rhoKS = 100.0, safety_factor = 1.5;
    auto ksfail = KSFailure<T, DeviceVec>(rhoKS, safety_factor);

    // compute the function values
    T mass_val = interface.evalFunction(mass);
    T ksfail_val = interface.evalFunction(ksfail);
    printf("mass %.2e, ksfail %.2e\n", mass_val, ksfail_val);

    // try solving an adjoint problem
    interface.evalFunction(ksfail);
    interface.solve_adjoint(ksfail);
    interface.writeAdjointSolution("out/plate_mg_adj1.vtk");

    // compute the design gradient
    // T *dptr = ksfail->dv_sens.getPtr();
    // T *h_dvgrad = new T[10];
    // hipMemcpy(h_dvgrad, dptr, 10 * sizeof(T), hipMemcpyDeviceToHost);
    // printf("h_dvgrad: ");
    // printVec<T>(10, h_dvgrad);

    // try setting design variables and solving again
    d_dvs.setFullVecToConstValue(thick * 2.0);
    interface.set_design_variables(d_dvs);
    interface.solve(d_loads);
    interface.writeSoln("out/plate_mg2.vtk");
}

int main(int argc, char **argv) {
    // input ----------
    bool is_multigrid = false;
    int nxe = 256; // default value
    // int nxe = 64;
    double SR = 100.0; // default
    int n_vcycles = 50;

    multigrid_plate_solve(nxe, SR, n_vcycles);

    return 0;

    
}