#include "hip/hip_runtime.h"
// general gpu_fem imports
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"
#include "mesh/TACSMeshLoader.h"
#include "mesh/vtk_writer.h"

// shell imports
#include "assembler.h"
#include "element/shell/shell_elem_group.h"
#include "element/shell/physics/isotropic_shell.h"

// local multigrid imports
#include "multigrid/grid.h"
#include "multigrid/fea.h"
#include "multigrid/solvers/gmg.h"
#include <string>
#include <chrono>

// new multigrid imports for K-cycles, etc.
#include "multigrid/solvers/solve_utils.h"
#include "multigrid/solvers/direct/cusp_directLU.h"
#include "multigrid/solvers/krylov/bsr_pcg.h"
#include "multigrid/solvers/multilevel/kcycle.h"
#include "multigrid/solvers/multilevel/twolevel.h"

/* command line args:
    [direct/mg] [--nxe int] [--SR float] [--nvcyc int]
    * nxe must be power of 2

    examples:
    ./1_plate.out direct --nxe 2048 --SR 100.0    to run direct plate solve on 2048 x 2048 elem grid with slenderness ratio 100
    ./1_plate.out mg --nxe 2048 --SR 100.0    to run geometric multigrid plate solve on 2048 x 2048 elem grid with slenderness ratio 100
*/

void to_lowercase(char *str) {
    for (; *str; ++str) {
        *str = std::tolower(*str);
    }
}

void direct_plate_solve(int nxe, double SR) {
    using T = double;   
    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;

    constexpr bool has_ref_axis = false;
    constexpr bool is_nonlinear = false;
    using Data = ShellIsotropicData<T, has_ref_axis>;
    using Physics = IsotropicShell<T, Data, is_nonlinear>;

    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
    using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

    auto start0 = std::chrono::high_resolution_clock::now();

    int nye = nxe;
    double Lx = 1.0, Ly = 1.0, E = 70e9, nu = 0.3, thick = 1.0 / SR, rho = 2500, ys = 350e6;
    int nxe_per_comp = nxe / 4, nye_per_comp = nye/4; // for now (should have 25 grids)
    auto assembler = createPlateAssembler<Assembler>(nxe, nye, Lx, Ly, E, nu, thick, rho, ys, nxe_per_comp, nye_per_comp);

    // BSR symbolic factorization
    // must pass by ref to not corrupt pointers
    auto& bsr_data = assembler.getBsrData();
    double fillin = 10.0;  // 10.0
    bool print = true;
    bool full_LU = true;

    if (full_LU) {
        bsr_data.AMD_reordering();
        bsr_data.compute_full_LU_pattern(fillin, print);
    } else {
        /*
        RCM and reorderings actually hurt GMRES performance on the plate case
        because the matrix already has a nice banded structure => RCM increases bandwidth (which means it just doesn't work well for this problem
        as it's whole point is to decrease matrix bandwidth)
        */

        bsr_data.AMD_reordering();
        // bsr_data.RCM_reordering();
        // bsr_data.qorder_reordering(1.0);
        
        bsr_data.compute_ILUk_pattern(5, fillin);
        // bsr_data.compute_full_LU_pattern(fillin, print); // reordered full LU here for debug
    }
    // printf("perm:");
    // printVec<int>(bsr_data.nnodes, bsr_data.perm);
    assembler.moveBsrDataToDevice();

    // get the loads
    double Q = 1.0; // load magnitude
    // T *my_loads = getPlatePointLoad<T, Physics>(nxe, nye, Lx, Ly, Q);
    T *my_loads = getPlateLoads<T, Physics>(nxe, nye, Lx, Ly, Q);

    auto loads = assembler.createVarsVec(my_loads);
    assembler.apply_bcs(loads);

    // setup kmat and initial vecs
    auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);
    auto soln = assembler.createVarsVec();
    auto res = assembler.createVarsVec();
    auto vars = assembler.createVarsVec();

    // assemble the kmat
    assembler.add_jacobian(res, kmat);
    assembler.apply_bcs(res);
    assembler.apply_bcs(kmat);

    auto end0 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> startup_time = end0 - start0;
    CHECK_CUDA(hipDeviceSynchronize());
    auto start1 = std::chrono::high_resolution_clock::now();

    // solve the linear system
    if (full_LU) {
        CUSPARSE::direct_LU_solve(kmat, loads, soln);
    } else {
        int n_iter = 200, max_iter = 400;
        T abs_tol = 1e-11, rel_tol = 1e-14;
        bool print = true;
        CUSPARSE::GMRES_solve<T>(kmat, loads, soln, n_iter, max_iter, abs_tol, rel_tol, print);

        // CUSPARSE::GMRES_DR_solve<T, false>(kmat, loads, soln, 50, 10, 65, abs_tol, rel_tol, true, false, 5);
    }

    CHECK_CUDA(hipDeviceSynchronize());
    auto end1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> solve_time = end1 - start1;
    int nx = nxe + 1;
    int ndof = nx * nx * 6;
    double total = startup_time.count() + solve_time.count();
    size_t bytes_per_double = sizeof(double);
    double mem_mb = static_cast<double>(bytes_per_double) * static_cast<double>(bsr_data.nnzb) * 36.0 / 1024.0 / 1024.0;
    printf("plate direct solve, ndof %d : startup time %.2e, solve time %.2e, total %.2e, with mem (MB) %.2e\n", ndof, startup_time.count(), solve_time.count(), total, mem_mb);


    // print some of the data of host residual
    auto h_soln = soln.createHostVec();
    printToVTK<Assembler,HostVec<T>>(assembler, h_soln, "out/plate.vtk");
}

void multigrid_plate_solve(int nxe, double SR, int nsmooth, int ninnercyc, std::string cycle_type) {
    // geometric multigrid method here..
    // need to make a number of grids..

    using T = double;   
    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;
    constexpr bool has_ref_axis = false;
    constexpr bool is_nonlinear = false;
    using Data = ShellIsotropicData<T, has_ref_axis>;
    using Physics = IsotropicShell<T, Data, is_nonlinear>;
    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
    using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

    // multigrid objects
    // const SMOOTHER smoother = LEXIGRAPHIC_GS;
    // const SMOOTHER smoother = MULTICOLOR_GS;
    // const SMOOTHER smoother = MULTICOLOR_GS_FAST; 
    const SMOOTHER smoother = MULTICOLOR_GS_FAST2; // this is much faster than other two methods (MULTICOLOR_GS_FAST is about 2.6x slower at high DOF)
    // const SMOOTHER smoother = DAMPED_JACOBI;

    const SCALER scaler  = LINE_SEARCH;

    using Prolongation = StructuredProlongation<PLATE>;
    using GRID = ShellGrid<Assembler, Prolongation, smoother, scaler>;
    using MG = GeometricMultigridSolver<GRID>;

    // for K-cycles
    using DirectSolve = CusparseMGDirectLU<GRID>;
    using KrylovSolve = PCGSolver<T, GRID>;
    using TwoLevelSolve = MultigridTwoLevelSolver<GRID>;
    using KMG = MultilevelKcycleSolver<GRID, DirectSolve, TwoLevelSolve, KrylovSolve>;

    CHECK_CUDA(hipDeviceSynchronize());
    auto start0 = std::chrono::high_resolution_clock::now();
    
    MG *mg;
    KMG *kmg;

    bool is_kcycle = cycle_type == "K";
    if (is_kcycle) {
        kmg = new KMG();
    } else {
        mg = new MG();
    }

    // get nxe_min for not exactly power of 2 case
    int pre_nxe_min = nxe > 32 ? 32 : 4;
    int nxe_min = pre_nxe_min;
    for (int c_nxe = nxe; c_nxe >= pre_nxe_min; c_nxe /= 2) {
        nxe_min = c_nxe;
    }

    // int nxe_min = nxe / 2; // two level

    // make each grid
    for (int c_nxe = nxe; c_nxe >= nxe_min; c_nxe /= 2) {
        // make the assembler
        int c_nye = c_nxe;
        double Lx = 1.0, Ly = 1.0, E = 70e9, nu = 0.3, thick = 1.0 / SR, rho = 2500, ys = 350e6;
        int nxe_per_comp = c_nxe / 4, nye_per_comp = c_nye/4; // for now (should have 25 grids)
        auto assembler = createPlateAssembler<Assembler>(c_nxe, c_nye, Lx, Ly, E, nu, thick, rho, ys, nxe_per_comp, nye_per_comp);
        double Q = 1.0; // load magnitude
        T *my_loads = getPlateLoads<T, Physics>(c_nxe, c_nye, Lx, Ly, Q);
        printf("making grid with nxe %d\n", c_nxe);

        // make the grid
        bool full_LU = c_nxe == nxe_min; // smallest grid is direct solve
        bool reorder;
        if (smoother == LEXIGRAPHIC_GS) {
            reorder = false;
        } else if (smoother == MULTICOLOR_GS || smoother == MULTICOLOR_GS_FAST || smoother == MULTICOLOR_GS_FAST2) {
            reorder = true;
        } else if (smoother == DAMPED_JACOBI) {
            reorder = false;
        }
        auto grid = *GRID::buildFromAssembler(assembler, my_loads, full_LU, reorder);
        
        if (is_kcycle) {
            kmg->grids.push_back(grid);
        } else {
            mg->grids.push_back(grid);
        }
    }

    CHECK_CUDA(hipDeviceSynchronize());
    auto end0 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> startup_time = end0 - start0;

    T init_resid_nrm = is_kcycle ? kmg->grids[0].getResidNorm() : mg->grids[0].getResidNorm();

    CHECK_CUDA(hipDeviceSynchronize());
    auto start1 = std::chrono::high_resolution_clock::now();

    int pre_smooth = nsmooth, post_smooth = nsmooth; // need a little extra smoothing on cylinder (compare to plate).. (cause of curvature I think..)
    bool print = true;
    // bool print = false;
    T atol = 1e-6, rtol = 1e-6;

    T omega = 1.5;
    // T omega = 1.3;
    // T omega = 1.2; // worse than <1 for cylinder
    // T omega = 1.0;
    // T omega = 0.85; // a bit faster than 1.0 (and actually smooths it)
    // bool double_smooth = false;
    bool double_smooth = true; // twice as many smoothing steps at lower levels (similar cost, better conv?)

    int n_cycles = 500; // max # cycles
    int print_freq = 3;

    if (is_kcycle) {
        int n_krylov = 500;
        kmg->init_outer_solver(nsmooth, ninnercyc, n_krylov, omega, atol, rtol, print_freq, print);    
    }

    // fastest is K-cycle usually
    if (cycle_type == "V") {
        mg->vcycle_solve(0, pre_smooth, post_smooth, n_cycles, print, atol, rtol, omega, double_smooth, print_freq); //(good option)
    } else if (cycle_type == "W") {
        mg->wcycle_solve(0, pre_smooth, post_smooth, n_cycles, print, atol, rtol, omega);
    } else if (cycle_type == "F") {
        mg->fcycle_solve(0, pre_smooth, post_smooth, n_cycles, print, atol, rtol, omega, double_smooth, print_freq); // also decent
    } else if (cycle_type == "K") {
        kmg->solve(); // best
    }

    auto end1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> solve_time = end1 - start1;
    int ndof = cycle_type == "K" ? kmg->grids[0].N : mg->grids[0].N;
    double total = startup_time.count() + solve_time.count();
    double mem_MB = is_kcycle ? kmg->get_memory_usage_mb() : mg->get_memory_usage_mb();
    printf("plate GMG solve, ndof %d : startup time %.2e, solve time %.2e, total %.2e, with mem(MB) %.2e\n", ndof, startup_time.count(), solve_time.count(), total, mem_MB);

    if (is_kcycle) {
        // print some of the data of host residual
        int *d_perm = kmg->grids[0].d_perm;
        auto h_soln = kmg->grids[0].d_soln.createPermuteVec(6, d_perm).createHostVec();
        printToVTK<Assembler,HostVec<T>>(kmg->grids[0].assembler, h_soln, "out/plate_mg.vtk");
    } else {
        // print some of the data of host residual
        int *d_perm = mg->grids[0].d_perm;
        auto h_soln = mg->grids[0].d_soln.createPermuteVec(6, d_perm).createHostVec();
        printToVTK<Assembler,HostVec<T>>(mg->grids[0].assembler, h_soln, "out/plate_mg.vtk");
    }
}

int main(int argc, char **argv) {
    // input ----------
    bool is_multigrid = false;
    int nxe = 256; // default value
    double SR = 100.0; // default
    int n_vcycles = 50;

    int nsmooth = 1; // typically faster right now
    int ninnercyc = 2; // inner V-cycles to precond K-cycle
    std::string cycle_type = "K"; // "V", "F", "W", "K"

    // Parse arguments
    for (int i = 1; i < argc; ++i) {
        char* arg = argv[i];
        to_lowercase(arg);

        if (strcmp(arg, "direct") == 0) {
            is_multigrid = false;
        } else if (strcmp(arg, "mg") == 0) {
            is_multigrid = true;
        } else if (strcmp(arg, "--nxe") == 0) {
            if (i + 1 < argc) {
                nxe = std::atoi(argv[++i]);
            } else {
                std::cerr << "Missing value for --nxe\n";
                return 1;
            }
        }  else if (strcmp(arg, "--sr") == 0) {
            if (i + 1 < argc) {
                SR = std::atof(argv[++i]);
            } else {
                std::cerr << "Missing value for --SR\n";
                return 1;
            }
        } else if (strcmp(arg, "--cycle") == 0) {
            if (i + 1 < argc) {
                cycle_type = argv[++i];
            } else {
                std::cerr << "Missing value for --level\n";
                return 1;
            }
        } else if (strcmp(arg, "--nsmooth") == 0) {
            if (i + 1 < argc) {
                nsmooth = std::atoi(argv[++i]);
            } else {
                std::cerr << "Missing value for --nsmooth\n";
                return 1;
            }
        } else if (strcmp(arg, "--ninnercyc") == 0) {
            if (i + 1 < argc) {
                ninnercyc = std::atoi(argv[++i]);
            } else {
                std::cerr << "Missing value for --nsmooth\n";
                return 1;
            }
        } else {
            std::cerr << "Unknown argument: " << argv[i] << std::endl;
            std::cerr << "Usage: " << argv[0] << " [direct/mg] [--nxe value] [--SR value] [--cycle char] [--nsmooth int] [--ninnercyc int]" << std::endl;
            return 1;
        }
    }

    // done reading arts, now run stuff
    if (is_multigrid) {
        multigrid_plate_solve(nxe, SR, nsmooth, ninnercyc, cycle_type);
    } else {
        direct_plate_solve(nxe, SR);
    }

    return 0;

    
}