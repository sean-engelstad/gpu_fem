#include "hip/hip_runtime.h"
#include "include/pde.h"

int main() {
    // make poisson solver and try and run it..

    using T = float; // double
    using GRID = PoissonSolver<T>;

    // set finest grid size
    // int nxe = 1024;
    int nxe = 32;

    int log2_nxe = 0, nxe_copy = nxe;
    while (nxe_copy >>= 1) ++log2_nxe;
    int n_levels = log2_nxe - 1;
    printf("nxe %d, log2(nxe) %d, n_levels %d\n", nxe, log2_nxe, n_levels);

    GRID *grids = new GRID[n_levels]; 
    int c_nxe = nxe;
    for (int ilevel = 0; ilevel < n_levels; ilevel++, c_nxe /= 2) {
        printf("level %d, making poisson solver with nxe %d elems\n", ilevel, c_nxe);
        grids[ilevel] = GRID(c_nxe);
    }

    /* DEBUG before full V-cycles */
    // T grid1_def_nrm = grids[n_levels-2].getDefectNorm();

    // // test restriction on coarsest grid
    // grids[n_levels-1].restrict_defect(grids[n_levels-2].d_defect);
    // // hipDeviceSynchronize();

    // T *h_coarse_defect0 = grids[n_levels-1].d_defect.createHostVec().getPtr();
    // printf("coarsest grid defect: ");
    // printVec<T>(grids[n_levels-1].N, h_coarse_defect0);

    // T grid2_def_nrm = grids[n_levels-1].getDefectNorm(); 

    // printf("grid 1 |defect| = %.2e => grid 2 |defect| = %.2e\n", grid1_def_nrm, grid2_def_nrm);

    // return 0;
    /* end of DEBUG section */

    // now try multigrid V-cycle solves here

    // /* try solve here.. */
    int pre_smooth = 3, post_smooth = 3;
    T omega = 2.0 / 3.0;
    bool print = true;
    // int n_vcycles = 100;
    int n_vcycles = 1;

    for (int i_vcycle = 0; i_vcycle < n_vcycles; i_vcycle++) {
        printf("V cycle step %d\n", i_vcycle);

        // go down each level smoothing and restricting until lowest level
        for (int i_level = 0; i_level < n_levels; i_level++) {
            // if not last  (pre-smooth)
            if (i_level < n_levels - 1) {
                printf("\tlevel %d pre-smooth\n", i_level);

                // pre-smooth
                grids[i_level].dampedJacobiDefect(pre_smooth, omega, print, pre_smooth - 1);

                // restrict defect
                grids[i_level + 1].restrict_defect(grids[i_level].d_defect);
            } else {
                printf("\tlevel %d full-solve\n", i_level);

                // print the defect here..
                // T *h_coarse_defect = grids[i_level].d_defect.createHostVec().getPtr();
                // printf("h_coarse_defect: ");
                // printVec<T>(grids[i_level].N, h_coarse_defect);

                // full-solve on last grid (of current defect)
                grids[i_level].dampedJacobiDefect(100, omega, print, 99);
            }
        }

        // now go back up the hierarchy
        for (int i_level = n_levels - 2; i_level >= 0; i_level--) {
            // get coarse-fine correction from coarser grid to this grid
            grids[i_level].prolongate(grids[i_level + 1].d_soln);

            printf("\tlevel %d post-smooth\n", i_level);

            // post-smooth
            grids[i_level].dampedJacobiDefect(post_smooth, omega, print, post_smooth - 1);
        }

        // compute fine grid defect of V-cycle
        T defect_nrm = grids[0].getDefectNorm();
        printf("\tend of v-cycle step %d, ||defect|| = %.2e\n", i_vcycle, defect_nrm);

    }

    // free
    for (int ilevel = 0; ilevel < n_levels; ilevel++) {
        grids[ilevel].free();
    }

    return 0;
};