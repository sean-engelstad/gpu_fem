#include "hip/hip_runtime.h"
#include "../tests/test_commons.h"
#include "chrono"
#include "linalg/_linalg.h"

// shell imports for local performance optimization
#include "include/v1/v1.h"
#include "include/v3/v3.h"

template <typename T, class ElemGroup, class Data, bool simple = false>
__global__ void test_drill_strain_kernel() {

    int iquad = threadIdx.x;
    int inode = iquad;
    Data physData{7e9, 0.3, 1e-2};

    T xpts[12], vars[24];
    for (int i = 0; i < 12; i++) {
        xpts[i] = 0.123 + 0.5234 * i + 0.00123 * i * i;
    }
    for (int i = 0; i < 24; i++) {
        vars[i] = -0.123 + 0.5234 * i - 0.00123 * i * i;
    }

    // XdinvT, Tmat
    T Tmatn[36], XdinvTn[36], detXdq;
    bool active_thread = true;
    ElemGroup::template compute_nodal_transforms<Data>(active_thread, inode, xpts, physData, &Tmatn[9 * inode], &XdinvTn[9 * inode]);
    ElemGroup::template compute_quadpt_transforms<Data>(active_thread, iquad, xpts, &detXdq);

    // if (threadIdx.x == 0) {
    //     printf("Tmatn:");
    //     printVec<T>(9, &Tmatn[9 * inode]);
    //     printf("XdinvTn:");
    //     printVec<T>(9, &XdinvTn[9 * inode]);
    // }

    if constexpr (!simple) {
        // tranpose Tmatn, XdinvTn in place
        for (int inode = 0; inode < 4; inode++) {
            T *_Tmatn = &Tmatn[9 * inode];
            T *_XdinvTn = &XdinvTn[9 * inode];
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < i; j++) {
                    T swap = _Tmatn[3 * i + j];
                    _Tmatn[3 * i + j] = _Tmatn[3 * j + i];
                    _Tmatn[3 * j + i] = swap;

                    swap = _XdinvTn[3 * i + j];
                    _XdinvTn[3 * i + j] = _XdinvTn[3 * j + i];
                    _XdinvTn[3 * j + i] = swap;
                }
            }
        }
    }

    // test for single element
    T local_res[24];
    memset(local_res, 0.0, 24 * sizeof(T));

    ElemGroup::template add_drill_strain_quadpt_residual_fast<Data, simple>(active_thread, iquad, vars, physData, &Tmatn[9 * inode], 
        &XdinvTn[9 * inode], detXdq, local_res);

    if (threadIdx.x == 0) {
        printf("local_res:");
        printVec<T>(6, local_res);
    }
}

template <typename T, class ElemGroup1, class Data>
void test_drill_strain_resid_ref() {
    T resid1[24];
    for (int i = 0; i < 24; i++) {
        resid1[i] = 0.0;
    }

    Data physData{7e9, 0.3, 1e-2};

    T xpts[12], vars[24];
    for (int i = 0; i < 12; i++) {
        xpts[i] = 0.123 + 0.5234 * i + 0.00123 * i * i;
    }
    for (int i = 0; i < 24; i++) {
        vars[i] = -0.123 + 0.5234 * i - 0.00123 * i * i;
    }

    // original resid computation with debug_mode = 1 for drill strains
    constexpr int debug_mode = 1;
    for (int iquad = 0; iquad < 1; iquad++) {
        ElemGroup1::template add_element_quadpt_residual<Data, debug_mode>(true, iquad, xpts, vars,
                                                                       physData, resid1);
    }

    printf("resid_ref:");
    printVec<T>(6, resid1);
}

template <typename T, int vars_per_node, class Data, class Basis, class Director>
void test_drill_strain_fwd(const T quad_pt[], const Data &physData, const T xpts[],
                           const T vars[]) {
    T et1;

    // original fwd
    T fn[12];
    ShellComputeNodeNormals<T, Basis>(xpts, fn);
    ShellComputeDrillStrain<T, vars_per_node, Data, Basis, Director>(quad_pt, physData.refAxis,
                                                                     xpts, vars, fn, &et1);
    T et2 = 5.50449708e-01;
    printf("\tet1 = %.8e\n", et1);
}

void test_v1() {
    using T = double;
    constexpr bool is_nonlinear = true; // true
    using Quad = QuadLinearQuadratureV1<T>;
    using Director = LinearizedRotationV1<T>;
    using Basis = ShellQuadBasisV1<T, Quad, 2>;
    using Data = ShellIsotropicDataV1<T, false>;
    using Physics = IsotropicShellV1<T, Data, is_nonlinear>;
    using ElemGroup1 = ShellElementGroupV1<T, Director, Basis, Physics>;

    Data physData{7e9, 0.3, 1e-2};

    T xpts[12], vars[24];
    for (int i = 0; i < 12; i++) {
        xpts[i] = 0.123 + 0.5234 * i + 0.00123 * i * i;
    }
    for (int i = 0; i < 24; i++) {
        vars[i] = -0.123 + 0.5234 * i - 0.00123 * i * i;
    }

    T quad_pt[2];
    Quad::getQuadraturePoint(0, quad_pt);

    // test_drill_strain_fwd<T, 6, Data, Basis, Director>(quad_pt, physData, xpts, vars);
    test_drill_strain_resid_ref<T, ElemGroup1, Data>();
}

void test_v3() {
    using T = double;
    constexpr bool is_nonlinear = true; // true
    constexpr int kernel_option = 4;

    using Quad = QuadLinearQuadratureV3<T>;
    using Director = LinearizedRotationV3<T>;
    using Basis = ShellQuadBasisV3<T, Quad>;
    using Data = ShellIsotropicDataV3<T, false>;
    using Physics = IsotropicShellV3<T, Data, is_nonlinear>;
    using ElemGroup = ShellElementGroupV3<T, Director, Basis, Physics, kernel_option>; //, full_strain>;
    using Assembler = ElementAssemblerV3<T, ElemGroup, VecType, BsrMat>;

    constexpr int n_quad = 4; // 4
    test_drill_strain_kernel<T, ElemGroup, Data, false><<<1,n_quad>>>();
    test_drill_strain_kernel<T, ElemGroup, Data, true><<<1,n_quad>>>();
    CHECK_CUDA(hipDeviceSynchronize());
}


int main() {

    printf("\n");
    test_v1();
    printf("-------\n");
    test_v3();

    return 0;
}