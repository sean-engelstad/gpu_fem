#include "hip/hip_runtime.h"
#include "../../examples/plate/_plate_utils.h"
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"
#include "mesh/TACSMeshLoader.h"
#include "mesh/vtk_writer.h"
#include "../test_commons.h"
#include <cassert>
#include <string>
#include <list>

// cusparse and cublas
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"

// shell imports
#include "assembler.h"
#include "element/shell/shell_elem_group.h"
#include "element/shell/physics/isotropic_shell.h"


void test_mat_vec_product(std::string ordering, std::string fill_type, bool print = false) {
    // bool print = false;
    // std::string ordering = argv[1];   // "none", "RCM", or "qorder"
    // std::string fill_type = argv[2];  // "nofill", "ILUk", or "LU"

    int rcm_iters = 5;
    double p_factor = 1.0;
    int k = 1; // for ILU(k)
    int nxe = 2;
    double fillin = 10.0;

    // ----------------------------------

    using T = double;   

    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;

    constexpr bool has_ref_axis = false;
    constexpr bool is_nonlinear = false;
    using Data = ShellIsotropicData<T, has_ref_axis>;
    using Physics = IsotropicShell<T, Data, is_nonlinear>;

    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
    using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;
    int nye = nxe;
    double Lx = 2.0, Ly = 1.0, E = 70e9, nu = 0.3, thick = 0.005;
    auto assembler = createPlateAssembler<Assembler>(nxe, nye, Lx, Ly, E, nu, thick);

    // first compute Kmat with no permutation and no fillin
    // ----------------------------------------------------

    // get bsr data and make copy of original for later
    auto& bsr_data0 = assembler.getBsrData();
    auto bsr_data = bsr_data0.createDeviceBsrData().createHostBsrData();

    // Apply fill pattern
    if (fill_type == "nofill") {
        bsr_data0.compute_nofill_pattern();
    } else if (fill_type == "ILUk") {
        bsr_data0.compute_ILUk_pattern(k, fillin, print);
    } else if (fill_type == "LU") {
        bsr_data0.compute_full_LU_pattern(fillin);
    } else {
        std::cerr << "Unknown fill type: " << fill_type << "\n";
        return;
    }

    // get bsr data on host after fillin for later kmat comprison check
    auto h_bsr_data_orig = bsr_data0.createDeviceBsrData().createHostBsrData();
    auto d_bsr_data_orig = bsr_data0.createDeviceBsrData(); // have to detach from ref to assembler
    assembler.moveBsrDataToDevice();

    // assemble unpermuted kmat
    auto kmat0 = createBsrMat<Assembler, VecType<T>>(assembler);
    auto res = assembler.createVarsVec();
    assembler.add_jacobian(res, kmat0);

    // get values off the device
    auto kmat0_vals = kmat0.getVec().createHostVec();

    // if (print) printf("\n\n\n--------------------------------\ndone with first assembly\n--------------------------------\n\n\n");

    // then compute Kmat = P * Kmat * P^Twith permutation and no fillin
    // ----------------------------------------------------------------

    // reorder the bsr data
    if (ordering == "RCM") {
        bsr_data.RCM_reordering(rcm_iters);
    } else if (ordering == "AMD") {
        bsr_data.AMD_reordering();
    } else if (ordering == "qorder") {
        bsr_data.qorder_reordering(p_factor, rcm_iters, print);
    } else if (ordering != "none") {
        std::cerr << "Unknown ordering: " << ordering << "\n";
        return;
    }

    // compute nofill and set new bsr data into it on the device
    // Apply fill pattern
    if (fill_type == "nofill") {
        bsr_data.compute_nofill_pattern();
    } else if (fill_type == "ILUk") {
        bsr_data.compute_ILUk_pattern(k, fillin, print);
    } else if (fill_type == "LU") {
        bsr_data.compute_full_LU_pattern(fillin);
    } else {
        std::cerr << "Unknown fill type: " << fill_type << "\n";
        return;
    }

    auto d_bsr_data = bsr_data.createDeviceBsrData();
    assembler.setBsrData(d_bsr_data);

    // assemble permuted kmat (nofill)
    auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);
    assembler.add_jacobian(res, kmat);

    // get values off the device
    auto kmat_vals = kmat.getVec().createHostVec();

    // check kmat error again for debugging
    // ------------------------------------

    bool sparsity_pass = true; // sparsity pass should only be checked for nofill
    double K_abs_err = 0.0;
    int block_dim = bsr_data.block_dim;
    int block_dim2 = block_dim * block_dim;

    for (int i = 0; i < bsr_data.nnodes; i++) {
        int i2 = bsr_data.iperm[i]; // old row => new row with iperm

        // loop through original cols
        for (int jp = h_bsr_data_orig.rowp[i]; jp < h_bsr_data_orig.rowp[i+1]; jp++) {
            int j = h_bsr_data_orig.cols[jp];
            

            // loop through new cols to find match
            int jp2 = -1, j2 = -1;
            for (int _jp2 = bsr_data.rowp[i2]; _jp2 < bsr_data.rowp[i2+1]; _jp2++) {
                int _j2 = bsr_data.cols[_jp2]; // new cols
                if (bsr_data.iperm[j] == _j2) { // iperm[old col] == new col
                    jp2 = _jp2;
                    j2 = _j2;
                    break;
                }
            }
            // check found matching column, sparsity in agreement
            if (jp2 == -1 || j2 == -1) {
                sparsity_pass = false;
                if (print) printf("orig brow, bcol = %d, %d don't have matching sparsity\n", i, j);
            } else {
                // compute abs err among K values now, looping through the block
                for (int ii = 0; ii < block_dim2; ii++) {
                    double val0 = kmat0_vals[block_dim2 * jp + ii];
                    double valp = kmat_vals[block_dim2 * jp2 + ii];
                    // int ind1 = block_dim2 * jp + ii;
                    // int ind2 = block_dim2 * jp2 + ii;
                    // if (ind1 != ind2) {
                    //     printf("in K_abs_err, this ind not equal %d, %d\n", ind1, ind2);
                    // }
                    double c_abs_err = abs_err(val0, valp);
                    K_abs_err = max(K_abs_err, c_abs_err);
                }
            }
        }
    }

    if (print) {
        printf("sparsity pass %d, K_abs_err %.4e\n", sparsity_pass, K_abs_err);
    }

    // build unreordered and reordered vec
    // -----------------------------------

    // unreordered random vector
    int nvars = assembler.get_num_vars();
    std::random_device rd;  // seed source
    std::mt19937 gen(rd()); // Mersenne Twister engine
    std::uniform_real_distribution<> dis(0.0, 1.0);
    std::vector<T> test_vec(nvars);
    for (auto& val : test_vec) {
        val = dis(gen);
    }
    HostVec<T> h_test_vec(nvars, test_vec.data());
    auto d_test_vec = h_test_vec.createDeviceVec();

    // reordered random vector, iperm: old to new vals
    auto d_test_vec_perm = h_test_vec.createDeviceVec();
    d_test_vec_perm.permuteData(bsr_data.block_dim, d_bsr_data.iperm);

    // compare kmat*u before and after reordering
    // ------------------------------------------

    // create temp vecs for the products
    auto loads0 = assembler.createVarsVec();
    auto loads_perm = assembler.createVarsVec();

    // initial data needed for cusparse matrices
    int mb = bsr_data.nnodes; //, block_dim = bsr_data.block_dim;
    T *d_vals0 = kmat0.getPtr();
    T *d_vals_perm = kmat.getPtr();
    int *d_rowp0 = d_bsr_data_orig.rowp;
    int *d_cols0 = d_bsr_data_orig.cols;
    int *d_rowp_perm = d_bsr_data.rowp;
    int *d_cols_perm = d_bsr_data.cols;
    T *d_loads0 = loads0.getPtr();
    T *d_loads_perm = loads_perm.getPtr();
    T *d_u0 = d_test_vec.getPtr(); // final device pointer
    T *d_u_perm = d_test_vec_perm.getPtr();

    // create inital cublas and cusparse handles --------

    /* Create CUBLAS context */
    hipblasHandle_t cublasHandle = NULL;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    /* Create CUSPARSE context */
    hipsparseHandle_t cusparseHandle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    /* Description of the A matrix */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    // perform Kmat*test_vec unreordered
    T a = 1.0, b = 0.0;
    CHECK_CUSPARSE(hipsparseDbsrmv(
        cusparseHandle, 
        HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        mb, mb, d_bsr_data_orig.nnzb,
        &a, descrA,
        d_vals0, d_rowp0, d_cols0,
        block_dim,
        d_u0,
        &b,
        d_loads0
    ));

    // perform Kmat*test_vec reordered
    CHECK_CUSPARSE(hipsparseDbsrmv(
        cusparseHandle, 
        HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        mb, mb, bsr_data.nnzb,
        &a, descrA,
        d_vals_perm, d_rowp_perm, d_cols_perm,
        block_dim,
        d_u_perm,
        &b,
        d_loads_perm
    ));

    // use perm : new rows => old rows on d_temp_perm
    loads_perm.permuteData(bsr_data.block_dim, d_bsr_data.perm);

    // offload from the device -----------------------

    /* copy device vecs to the host */
    auto h_loads0 = loads0.createHostVec();
    auto h_loads_perm = loads_perm.createHostVec();

    // final test result -----------------------------

    // test name
    std::string testName = "Kmat*u reordering consistency test, with ";

    testName += ordering;  // always include the ordering name

    if (fill_type == "ILUk") {
        testName += " ILU(" + std::to_string(k) + ")";
    } else if (fill_type == "nofill") {
        testName += " nofill";
    } else if (fill_type == "LU") {
        testName += " LU";
    }

    // now print out test report
    double mat_vec_err = rel_err(h_loads0, h_loads_perm);
    bool pass = mat_vec_err < 1e-5;
    printTestReport(testName, pass, mat_vec_err);
}

int main() {
    // turn off test all for debugging
    bool test_all = true;

    if (test_all) {
        std::list<std::string> list1 = {"none", "RCM", "AMD", "qorder"};
        std::list<std::string> list2 = {"nofill", "ILUk", "LU"};

        for (auto it2 = list2.begin(); it2 != list2.end(); ++it2) {
            for (auto it1 = list1.begin(); it1 != list1.end(); ++it1) {
                test_mat_vec_product(*it1, *it2);
            }
        }
    } else {
        // test single failing test
        test_mat_vec_product("RCM", "nofill", true);
    }  
};