#include "hip/hip_runtime.h"
#include "utils/_laplace_utils.h"
#include "linalg/_linalg.h"
#include "solvers/linear_static_cusparse.h"
#include "../test_commons.h"

int main() {
    // double BSR mv routine doesn't work (see archive)
    // so need to use float instead for BSR matrix
    using T = double;

    // true solution for N=64 from python solver
    T true_soln[] = {0.48629073, 0.66913706, 0.74475527, 0.77346185, 0.77346185,
        0.74475527, 0.66913706, 0.48629073, 0.27602587, 0.44550224,
        0.53642216, 0.57563028, 0.57563028, 0.53642216, 0.44550224,
        0.27602587, 0.17231052, 0.30042386, 0.37980085, 0.41700683,
        0.41700683, 0.37980085, 0.30042386, 0.17231052, 0.11279234,
        0.20408183, 0.26535057, 0.29558935, 0.29558935, 0.26535057,
        0.20408183, 0.11279234, 0.07477702, 0.13776056, 0.18193023,
        0.20441065, 0.20441065, 0.18193024, 0.13776056, 0.07477702,
        0.04855519, 0.09025316, 0.12019915, 0.13571237, 0.13571237,
        0.12019915, 0.09025316, 0.04855519, 0.02919055, 0.05449776,
        0.07290081, 0.08252732, 0.08252732, 0.07290082, 0.05449776,
        0.02919055, 0.01370927, 0.02564651, 0.03437903, 0.03896878,
        0.03896878, 0.03437903, 0.02564651, 0.01370927};

    // case inputs
    // -----------
    // int N = 16384;
    int N = 64; // 16384
    int n_iter = min(N, 200);
    int max_iter = min(N, 200);
    constexpr bool use_precond = true;
    T abs_tol = 1e-7, rel_tol = 1e-8;
    bool print = true;

    // NOTE : starting with BSR matrix of block size 1 (just to demonstrate the correct cusparse methods for BSR)

    // initialize data
    // ---------------
    
    int *csr_rowp, *csr_cols;
    // int M = N;
    T *csr_vals, *rhs, *x;
    int nz = 5 * N - 4 * (int)sqrt((double)N);

    // allocate rowp, cols on host
    csr_rowp = (int*)malloc(sizeof(int) * (N + 1));
    csr_cols = (int*)malloc(sizeof(int) * nz);
    csr_vals = (T*)malloc(sizeof(T) * nz);
    x = (T*)malloc(sizeof(T) * N);
    rhs = (T*)malloc(sizeof(T) * N);

    for (int i = 0; i < N; i++) {
        x[i] = 0.0;
        rhs[i] = 0.0;    
    }

    // initialize data
    genLaplaceCSR<T>(csr_rowp, csr_cols, csr_vals, N, nz, rhs);
    // now rhs is not zero

    // convert to BSR
    int *rowp, *cols, nnzb;
    T *vals;
    int mb = N /2;
    int block_dim = 2;
    CSRtoBSR<T>(block_dim, N, csr_rowp, csr_cols, csr_vals, &rowp, &cols, &vals, &nnzb);

    // transfer data to the device
    int *d_rowp, *d_cols;
    T *d_vals, *d_x, *d_rhs;
    CHECK_CUDA(hipMalloc((void **)&d_rowp, (N/2+1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_cols, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_vals, 4 * nnzb * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_x, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void **)&d_rhs, N * sizeof(T)));

    // copy data for the matrix over to device
    CHECK_CUDA(hipMemcpy(d_rowp, rowp, (N/2+1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_cols, cols, nnzb * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_vals, vals, 4 * nnzb * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, x, N * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_rhs, rhs, N * sizeof(T), hipMemcpyHostToDevice));

    // make perm and iperm on device
    int *perm, *iperm;
    perm = new int[mb];
    iperm = new int[mb];
    for (int i = 0; i < mb; i++) {
        perm[i] = i;
        iperm[i] = i;
    }
    int *d_perm = HostVec<int>(mb, perm).createDeviceVec().getPtr();
    int *d_iperm = HostVec<int>(mb, iperm).createDeviceVec().getPtr();

    // compare these results with cuda_examples/gmres/3_gmres_Dbsr.cu
    // now make BSRData and BSRMat objects here
    auto bsr_data = BsrData(mb, block_dim, nnzb, d_rowp, d_cols, d_perm, d_iperm, false);
    auto vals_vec = DeviceVec<T>(4 * nnzb, d_vals);
    auto mat = BsrMat<DeviceVec<T>>(bsr_data, vals_vec);
    auto rhs_vec = DeviceVec<T>(N, d_rhs);
    auto soln_vec = DeviceVec<T>(N, d_x);

    bool debug = true;
    int print_freq = 3;

    // now call GMRES algorithm
    CUSPARSE::HGMRES_solve<T, use_precond>(mat, rhs_vec, soln_vec, n_iter, max_iter, abs_tol, rel_tol, print, debug, print_freq);

    // now check soln error?
    T max_rel_err = rel_err(soln_vec, true_soln);
    bool passed = EXPECT_VEC_NEAR(soln_vec, true_soln, 1e-6);
    printTestReport("GMRES N=16 Laplace test", passed, max_rel_err);
}