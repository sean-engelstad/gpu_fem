#include "hip/hip_runtime.h"
#include "../../examples/plate/_plate_utils.h"
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"
#include "mesh/TACSMeshLoader.h"
#include "mesh/vtk_writer.h"
#include "../test_commons.h"
#include <cassert>


// shell imports
#include "assembler.h"
#include "element/shell/shell_elem_group.h"
#include "element/shell/physics/isotropic_shell.h"

int main(int argc, char* argv[]) {
    using T = double;   

    bool reorder = true;
    int N = 30;
    
    if (argc == 1) {
        reorder = 
    }
    = argv[2]; // "nofill", "ILUk", or "LU"

    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;

    constexpr bool has_ref_axis = false;
    constexpr bool is_nonlinear = false;
    using Data = ShellIsotropicData<T, has_ref_axis>;
    using Physics = IsotropicShell<T, Data, is_nonlinear>;

    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
    using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

    // int nxe = 3;
    int nxe = 100;
    // int nxe = 300;
    int nye = nxe;
    double Lx = 2.0, Ly = 1.0, E = 70e9, nu = 0.3, thick = 0.005;
    auto assembler = createPlateAssembler<Assembler>(nxe, nye, Lx, Ly, E, nu, thick);

    // BSR symbolic factorization
    // must pass by ref to not corrupt pointers
    auto& bsr_data = assembler.getBsrData();
    double fillin = 10.0;  // 10.0
    bool print = true;
    bsr_data.AMD_reordering();
    bsr_data.compute_full_LU_pattern(fillin, print);
    assembler.moveBsrDataToDevice();

    // get the loads
    double Q = 1.0; // load magnitude
    // T *my_loads = getPlatePointLoad<T, Physics>(nxe, nye, Lx, Ly, Q);
    T *my_loads = getPlateLoads<T, Physics>(nxe, nye, Lx, Ly, Q);
    auto loads = assembler.createVarsVec(my_loads);
    assembler.apply_bcs(loads);

    // setup kmat and initial vecs
    auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);
    auto soln = assembler.createVarsVec();
    auto res = assembler.createVarsVec();
    auto vars = assembler.createVarsVec();

    // assemble the kmat
    assembler.add_jacobian(res, kmat);
    assembler.apply_bcs(res);
    assembler.apply_bcs(kmat);

    // solve the linear system
    CUSPARSE::direct_LU_solve(kmat, loads, soln);

    // print some of the data of host residual
    auto h_soln = soln.createHostVec();
    printToVTK<Assembler,HostVec<T>>(assembler, h_soln, "plate.vtk");

    // check the residual of the system
    assembler.set_variables(soln);
    assembler.add_residual(res); // internal residual
    auto rhs = assembler.createVarsVec();
    CUBLAS::axpy(1.0, loads, rhs);
    CUBLAS::axpy(-1.0, res, rhs); // rhs = loads - f_int
    assembler.apply_bcs(rhs);
    double resid_norm = CUBLAS::get_vec_norm(rhs);
    printf("resid_norm = %.4e\n", resid_norm);

    bool passed = abs(resid_norm) < 1e-6;
    printTestReport("direct LU plate solve", passed, resid_norm);
};