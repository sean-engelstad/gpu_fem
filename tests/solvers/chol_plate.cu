#include "hip/hip_runtime.h"
#include "../../examples/plate/_plate_utils.h"
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"
#include "mesh/TACSMeshLoader.h"
#include "mesh/vtk_writer.h"
#include "../test_commons.h"
#include <cassert>
#include <string>
#include <list>

// shell imports
#include "assembler.h"
#include "element/shell/shell_elem_group.h"
#include "element/shell/physics/isotropic_shell.h"

void test_chol_plate(std::string ordering, bool print = false, int nxe = 50) {
    using T = double;  

    double fillin = 10.0;
    int rcm_iters = 5;
    double p_factor = 1.0;

    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;

    constexpr bool has_ref_axis = false;
    constexpr bool is_nonlinear = false;
    using Data = ShellIsotropicData<T, has_ref_axis>;
    using Physics = IsotropicShell<T, Data, is_nonlinear>;

    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
    using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;
    int nye = nxe;
    double Lx = 2.0, Ly = 1.0, E = 70e9, nu = 0.3, thick = 0.005;
    auto assembler = createPlateAssembler<Assembler>(nxe, nye, Lx, Ly, E, nu, thick);

    // BSR symbolic factorization
    // must pass by ref to not corrupt pointers
    auto& bsr_data = assembler.getBsrData();
    if (ordering == "RCM") {
        bsr_data.RCM_reordering(rcm_iters);
    } else if (ordering == "AMD") {
        bsr_data.AMD_reordering();
    } else if (ordering == "qorder") {
        bsr_data.qorder_reordering(p_factor, rcm_iters, print);
    } else if (ordering != "none") {
        std::cerr << "Unknown ordering: " << ordering << "\n";
        return;
    }
    bsr_data.compute_full_LU_pattern(fillin, print);
    assembler.moveBsrDataToDevice();

    // get the loads
    double Q = 1.0; // load magnitude
    // T *my_loads = getPlatePointLoad<T, Physics>(nxe, nye, Lx, Ly, Q);
    T *my_loads = getPlateLoads<T, Physics>(nxe, nye, Lx, Ly, Q);
    auto loads = assembler.createVarsVec(my_loads);
    assembler.apply_bcs(loads);

    // setup kmat and initial vecs
    auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);
    auto soln = assembler.createVarsVec();
    auto res = assembler.createVarsVec();
    auto vars = assembler.createVarsVec();

    // assemble the kmat
    assembler.add_jacobian(res, kmat);
    assembler.apply_bcs(res);
    assembler.apply_bcs(kmat);

    // switch sparsity and values to cholesky
    kmat.switch_to_cholesky();

    // solve the linear system
    CUSPARSE::direct_cholesky_solve(kmat, loads, soln);

    // print some of the data of host residual
    auto h_soln = soln.createHostVec();
    printToVTK<Assembler,HostVec<T>>(assembler, h_soln, "plate.vtk");

    // check the residual of the system
    assembler.set_variables(soln);
    assembler.add_residual(res); // internal residual
    auto rhs = assembler.createVarsVec();
    CUBLAS::axpy(1.0, loads, rhs);
    CUBLAS::axpy(-1.0, res, rhs); // rhs = loads - f_int
    assembler.apply_bcs(rhs);
    double resid_norm = CUBLAS::get_vec_norm(rhs);
    if (print) printf("resid_norm = %.4e\n", resid_norm);

    // test report
    std::string testName = "direct Chol plate solve, with ";
    testName += ordering;

    bool passed = abs(resid_norm) < 1e-6;
    printTestReport(testName, passed, resid_norm);
}

int main(int argc, char* argv[]) {
    bool test_all = false;

    bool print = false;
    int nxe = 20;
    if (test_all) {
        std::list<std::string> list1 = {"none", "RCM", "AMD", "qorder"};

        for (auto it1 = list1.begin(); it1 != list1.end(); ++it1) {
            test_chol_plate(*it1, print, nxe);
        }
    } else {
        // test single failing test
        print = true;
        nxe = 20;
        test_chol_plate("AMD", print, nxe);
    }  
};