#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>

#include "chrono"
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"
#include "../test_commons.h"

// shell imports
#include "assembler.h"
#include "element/shell/physics/isotropic_shell.h"
#include "element/shell/shell_elem_group.h"

template <typename T, class Assembler>
HostVec<T> getTipLoads(Assembler &assembler, T length, T beam_tip_force) {
  // find nodes within tolerance of x=10.0
  int num_nodes = assembler.get_num_nodes();
  int num_vars = assembler.get_num_vars();
  HostVec<T> h_loads(num_vars);
  DeviceVec<T> d_xpts = assembler.getXpts();
  auto h_xpts = d_xpts.createHostVec();
  int num_tip_nodes = 0;
  for (int inode = 0; inode < num_nodes; inode++) {
    if (abs(h_xpts[3 * inode] - length) < 1e-6) {
      num_tip_nodes++;
    }
  }
  T nodal_force = beam_tip_force / num_tip_nodes;
  // printf("nodal force = %.4e\n", nodal_force);
  for (int inode = 0; inode < num_nodes; inode++) {
    if (abs(h_xpts[3 * inode] - length) < 1e-6) {
      h_loads[6 * inode + 2] = beam_tip_force / num_tip_nodes;
    }
  }
  return h_loads;
}

/**
 solve on CPU with cusparse for debugging
 **/
int main(void) {
  using T = double;
  bool print = false;

  std::ios::sync_with_stdio(false);  // always flush print immediately

  TACSMeshLoader<T> mesh_loader{};
  mesh_loader.scanBDFFile("baseline/Beam.bdf");

  using Quad = QuadLinearQuadrature<T>;
  using Director = LinearizedRotation<T>;
  using Basis = ShellQuadBasis<T, Quad, 2>;
  using Geo = Basis::Geo;

  constexpr bool has_ref_axis = false;
  constexpr bool is_nonlinear = true;
  // constexpr bool is_nonlinear = false;
  using Data = ShellIsotropicData<T, has_ref_axis>;
  using Physics = IsotropicShell<T, Data, is_nonlinear>;

  using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
  using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

  // material & thick properties
  double E = 1.2e6, nu = 0.0, thick = 0.1;
  auto assembler = Assembler::createFromBDF(mesh_loader, Data(E, nu, thick));

  // perform a factorization on the rowPtr, colPtr (before creating matrix)
  auto& bsr_data = assembler.getBsrData();
  // no fillin or reordering, let's just compute the kmat for comparison with TACS
  assembler.moveBsrDataToDevice();

  // setup kmat and initial vecs
  auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);
  auto res = assembler.createVarsVec();

  // assemble no fill kmat
  assembler.add_jacobian(res, kmat);
  assembler.apply_bcs(res);
  assembler.apply_bcs(kmat);

  // compare the global stiffness matrix
  auto h_kmat_vec = kmat.getVec().createHostVec();
  h_kmat_vec.print("kmat");

  // TODO : need CPU ref global kmat

  // T max_rel_err = rel_err(3, end_disp, ref_end_disp, 1e-8);
  T max_rel_err = 0.0;
  bool passed = max_rel_err < 1e-2;
  printTestReport("NL cantilever global kmat", passed, max_rel_err);

  return 0;
};