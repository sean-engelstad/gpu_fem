#include "assembler.h"
#include "shell/shell.h"
#include "chrono"

int main(void) {
    using T = double;

    using Quad = QuadLinearQuadrature<T>;
    using Director = LinearizedRotation<T>;
    using Basis = ShellQuadBasis<T, Quad, 2>;
    using Geo = Basis::Geo;

    constexpr bool has_ref_axis = false;
    using Data = ShellIsotropicData<T,has_ref_axis>;
    using Physics = IsotropicShell<T, Data>;
    
    using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>; 
    using Assembler = ElementAssembler<T, ElemGroup>;

    printf("running!\n");

    // int num_elements = 1;
    // int num_geo_nodes = 4;
    // int num_vars_nodes = 4;

    int num_geo_nodes = 1e2;
    int num_vars_nodes = 1e2;
    int num_elements = 1e3;

    // int num_geo_nodes = 3e3;
    // int num_vars_nodes = num_geo_nodes;
    // int num_elements = 1e5;

    // make fake element connectivity for testing
    int N = Geo::num_nodes * num_elements;
    int32_t *geo_conn = new int32_t[N];
    for (int i = 0; i < N; i++) {
      geo_conn[i] = i % num_geo_nodes; 
    }

    // randomly generate the connectivity for the variables / basis
    int N2 = Basis::num_nodes * num_elements;
    int32_t *vars_conn = new int32_t[N2];
    for (int i = 0; i < N2; i++) {
      vars_conn[i] = i % num_vars_nodes;
    }

    // set the xpts randomly for this example
    int32_t num_xpts = Geo::spatial_dim * num_geo_nodes;
    T *xpts = new T[num_xpts];
    for (int ixpt = 0; ixpt < num_xpts; ixpt++) {
      xpts[ixpt] = static_cast<double>(rand()) / RAND_MAX;
    }

    // initialize ElemData
    double E = 70e9, nu = 0.3, t = 0.005; // aluminum plate
    Data elemData[num_elements];
    for (int ielem = 0; ielem < num_elements; ielem++) {
        elemData[ielem] = Data(E, nu, t);
    }

    // make the assembler
    Assembler assembler(num_geo_nodes, num_vars_nodes, num_elements, geo_conn, vars_conn, xpts, elemData);

    // define variables here for testing different vars inputs
    // set some host data to zero
    int32_t num_vars = assembler.get_num_vars();
    T *h_vars = new T[num_vars];
    memset(h_vars, 0.0, num_vars * sizeof(T));

    bool nz_vars = true;
    if (nz_vars) {
      for (int ivar = 0; ivar < num_vars; ivar++) {
        h_vars[ivar] = static_cast<double>(rand()) / RAND_MAX;
      }
    }

    #ifdef USE_GPU
    T *d_vars;
    hipMalloc((void**)&d_vars, num_vars * sizeof(T));
    hipMemcpy(d_vars, h_vars, num_vars * sizeof(T), hipMemcpyHostToDevice);   
    assembler.set_variables(d_vars);
    #else // USE_GPU
    assembler.set_variables(h_vars);
    #endif
    

    // define the residual vector (host or device)
    T *h_residual = new T[num_vars];
    memset(h_residual, 0.0, num_vars * sizeof(T));
    #ifdef USE_GPU
    T *d_residual;
    hipMalloc((void**)&d_residual, num_vars * sizeof(T));
    hipMemset(d_residual, 0.0, num_vars * sizeof(T));     
    #endif

    int num_vars2 = num_vars*num_vars;
    T *h_mat = new T[num_vars2];
    memset(h_mat, 0.0, num_vars2 * sizeof(T));
    #ifdef USE_GPU
    T *d_mat;
    hipMalloc((void**)&d_mat, num_vars2 * sizeof(T));
    hipMemset(d_mat, 0.0, num_vars2 * sizeof(T));     
    #endif

    // time add residual method
    auto start = std::chrono::high_resolution_clock::now();

    // call add jacobian
    #ifdef USE_GPU
    assembler.add_jacobian(d_residual, d_mat);
    hipMemcpy(h_residual, d_residual, num_vars * sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy(h_mat, d_mat, num_vars * sizeof(T), hipMemcpyDeviceToHost);
    #else
    assembler.add_jacobian(h_residual, h_mat);
    #endif

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    // print data of host mat
    for (int i = 0; i < 24*24; i++) {
      printf("K[%d] = %.8e\n", i, h_mat[i]);
    }

    printf("took %d microseconds to run add jacobian\n", (int)duration.count());

    return 0;
};