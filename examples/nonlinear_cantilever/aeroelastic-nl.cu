#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>

#include "chrono"
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"
#include "coupled/_coupled.h"
// // #include "coupled/aero_solver.h"
// // #include "coupled/struct_solver.h"
// // #include "coupled/coupled_analysis.h"
// #include "coupled/meld.h"

// shell imports
#include "assembler.h"
#include "element/shell/physics/isotropic_shell.h"
#include "element/shell/shell_elem_group.h"

template <typename T, class Assembler>
HostVec<T> getTipLoads(Assembler &assembler, T length, T beam_tip_force) {
  // find nodes within tolerance of x=10.0
  int num_nodes = assembler.get_num_nodes();
  int num_vars = assembler.get_num_vars();
  HostVec<T> h_loads(num_vars);
  DeviceVec<T> d_xpts = assembler.getXpts();
  auto h_xpts = d_xpts.createHostVec();
  int num_tip_nodes = 0;
  for (int inode = 0; inode < num_nodes; inode++) {
    if (abs(h_xpts[3 * inode] - length) < 1e-6) {
      num_tip_nodes++;
    }
  }
  for (int inode = 0; inode < num_nodes; inode++) {
    if (abs(h_xpts[3 * inode] - length) < 1e-6) {
      h_loads[6 * inode + 2] = beam_tip_force / num_tip_nodes;
    }
  }
  return h_loads;
}

/**
 solve on CPU with cusparse for debugging
 **/
int main(void) {
  using T = double;

  std::ios::sync_with_stdio(false);  // always flush print immediately

  TACSMeshLoader<T> mesh_loader{};
  mesh_loader.scanBDFFile("Beam.bdf");

  using Quad = QuadLinearQuadrature<T>;
  using Director = LinearizedRotation<T>;
  using Basis = ShellQuadBasis<T, Quad, 2>;
  using Geo = Basis::Geo;

  constexpr bool has_ref_axis = false;
  constexpr bool is_nonlinear = true;
  // constexpr bool is_nonlinear = false;
  using Data = ShellIsotropicData<T, has_ref_axis>;
  using Physics = IsotropicShell<T, Data, is_nonlinear>;

  using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
  using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

  // define coupled analysis types
  // -----------------------------
  using StructSolver = TacsNonlinearStaticNewton<T, Assembler>;
  using AeroSolver = FixedAeroSolver<T, DeviceVec<T>>;
  using Transfer = MELD<T>;
  using CoupledDriver = FuntofemCoupledAnalysis<T, DeviceVec<T>, StructSolver, AeroSolver, Transfer>;


  // build the Tacs prelim objects
  // -----------------------------

  // material & thick properties
  double E = 1.2e6, nu = 0.0, thick = 0.1;
  auto assembler = Assembler::createFromBDF(mesh_loader, Data(E, nu, thick));

  // perform a factorization on the rowPtr, colPtr (before creating matrix)
  double fillin = 10.0;  // 10.0
  assembler.symbolic_factorization(fillin, true);

  // compute load magnitude of tip force
  double length = 10.0, width = 1.0;
  double Izz = width * thick * thick * thick / 12.0;
  double beam_tip_force = 4.0 * E * Izz / length / length;

  // compute loads
  auto h_loads = getTipLoads<T>(assembler, length, beam_tip_force);
  auto d_loads = h_loads.createDeviceVec();
  assembler.apply_bcs(d_loads);

  // setup kmat
  auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);
  auto linear_solve = CUSPARSE::direct_LU_solve<T>;
  
  // make the solvers and transfer scheme
  // ------------------------------------

  int num_load_factors = 20, num_newton = 30;
  StructSolver struct_solver = StructSolver(assembler, kmat, linear_solve, num_load_factors, num_newton);
  // make the struct linear solver
  // TacsLinearStatic struct_solver = TacsLinearStatic(assembler, kmat, linear_solve);  

  int na_surf = assembler.get_num_nodes();
  AeroSolver aero_solver = AeroSolver(na_surf, d_loads);

  auto d_xpts = assembler.getXpts(); // just use same mesh for aero and surf in this example
  T beta = 10.0, Hreg = 1e-4;
  int nn = 32, sym = -1;
  Transfer transfer = Transfer(d_xpts, d_xpts, beta, nn, sym, Hreg);
  transfer.initialize();

  // make coupled analysis object
  // ----------------------------

  int num_coupled_steps = 5;
  CoupledDriver driver = CoupledDriver(struct_solver, aero_solver, transfer, num_coupled_steps);
  driver.solve_forward();

  return 0;
};