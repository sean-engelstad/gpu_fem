#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>

#include "chrono"
#include "linalg/_linalg.h"
#include "solvers/_solvers.h"

// shell imports
#include "assembler.h"
#include "element/shell/physics/isotropic_shell.h"
#include "element/shell/shell_elem_group.h"

template <typename T, class Assembler>
HostVec<T> getTipLoads(Assembler &assembler, T length, T beam_tip_force) {
  // find nodes within tolerance of x=10.0
  int num_nodes = assembler.get_num_nodes();
  int num_vars = assembler.get_num_vars();
  HostVec<T> h_loads(num_vars);
  DeviceVec<T> d_xpts = assembler.getXpts();
  auto h_xpts = d_xpts.createHostVec();
  int num_tip_nodes = 0;
  for (int inode = 0; inode < num_nodes; inode++) {
    if (abs(h_xpts[3 * inode] - length) < 1e-6) {
      num_tip_nodes++;
    }
  }
  for (int inode = 0; inode < num_nodes; inode++) {
    if (abs(h_xpts[3 * inode] - length) < 1e-6) {
      h_loads[6 * inode + 2] = beam_tip_force / num_tip_nodes;
    }
  }
  return h_loads;
}

/**
 solve on CPU with cusparse for debugging
 **/
int main(void) {
  using T = double;

  std::ios::sync_with_stdio(false);  // always flush print immediately

  TACSMeshLoader<T> mesh_loader{};
  mesh_loader.scanBDFFile("Beam.bdf");

  using Quad = QuadLinearQuadrature<T>;
  using Director = LinearizedRotation<T>;
  using Basis = ShellQuadBasis<T, Quad, 2>;
  using Geo = Basis::Geo;

  constexpr bool has_ref_axis = false;
  constexpr bool is_nonlinear = true;
  // constexpr bool is_nonlinear = false;
  using Data = ShellIsotropicData<T, has_ref_axis>;
  using Physics = IsotropicShell<T, Data, is_nonlinear>;

  using ElemGroup = ShellElementGroup<T, Director, Basis, Physics>;
  using Assembler = ElementAssembler<T, ElemGroup, VecType, BsrMat>;

  // material & thick properties
  double E = 1.2e6, nu = 0.0, thick = 0.1;
  auto assembler = Assembler::createFromBDF(mesh_loader, Data(E, nu, thick));

  // perform a factorization on the rowPtr, colPtr (before creating matrix)
  double fillin = 10.0;  // 10.0
  assembler.symbolic_factorization(fillin, true);

  // compute load magnitude of tip force
  double length = 10.0, width = 1.0;
  double Izz = width * thick * thick * thick / 12.0;
  double beam_tip_force = 4.0 * E * Izz / length / length;

  // compute loads
  auto h_loads = getTipLoads<T>(assembler, length, beam_tip_force);
  auto d_loads = h_loads.createDeviceVec();
  assembler.apply_bcs(d_loads);

  // setup kmat, res, variables
  auto soln = assembler.createVarsVec();
  auto kmat = createBsrMat<Assembler, VecType<T>>(assembler);

  // newton solve
  int num_load_factors = 20, num_newton = 30;
  T min_load_factor = 0.05, max_load_factor = 1.0, abs_tol = 1e-8,
    rel_tol = 1e-5;
  auto solve_func = CUSPARSE::direct_LU_solve<T>;
  std::string outputPrefix = "out/beam_";
  newton_solve<T, BsrMat<DeviceVec<T>>, DeviceVec<T>, Assembler>(
      solve_func, kmat, d_loads, soln, assembler, num_load_factors,
      min_load_factor, max_load_factor, num_newton, abs_tol, rel_tol,
      outputPrefix);

  return 0;
};